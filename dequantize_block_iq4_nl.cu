#include "hip/hip_runtime.h"
template<typename dst_t>
static __global__ void dequantize_block_iq4_nl(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const auto i   = blockIdx.x;
    const block_iq4_nl * x = (const block_iq4_nl *) vx + i*(QK_K/QK4_NL);

    const auto tid = threadIdx.x;
    const int il = tid/8;
    const int ib = tid%8;
    dst_t * y = yy + i*QK_K + 32*ib + 4*il;
    const uint8_t  * q4 = x[ib].qs + 4*il;
    const float d = __half2float(x[ib].d);
    for (int j = 0; j < 4; ++j) {
        y[j+ 0] = d * kvalues_iq4nl[q4[j] & 0xf];
        y[j+16] = d * kvalues_iq4nl[q4[j] >>  4];
    }

}