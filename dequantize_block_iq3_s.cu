#include "hip/hip_runtime.h"
template<typename dst_t>
static __global__ void dequantize_block_iq3_s(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const auto i   = blockIdx.x;
    const block_iq3_s * x = (const block_iq3_s *) vx;

    const auto tid = threadIdx.x;
    const int il = tid/8;
    const int ib = tid%8;
    dst_t * y = yy + i*QK_K + 32*ib + 8*il;
    const uint8_t * qs = x[i].qs + 8*ib;
    const uint8_t * grid1 = (const uint8_t *)(iq3xs_grid + (qs[2*il+0] | ((x[i].qh[ib] << (8-2*il)) & 256)));
    const uint8_t * grid2 = (const uint8_t *)(iq3xs_grid + (qs[2*il+1] | ((x[i].qh[ib] << (7-2*il)) & 256)));
    const float d = __half2float(x[i].d) * (0.5f + ((x[i].scales[ib/2] >> 4*(ib%2)) & 0xf)) * 0.5f;
    const uint8_t signs = x[i].signs[4*ib + il];
    for (int j = 0; j < 4; ++j) {
        y[j+0] = d * grid1[j] * (signs & kmask_iq2xs[j+0] ? -1.f : 1.f);
        y[j+4] = d * grid2[j] * (signs & kmask_iq2xs[j+4] ? -1.f : 1.f);
    }
}