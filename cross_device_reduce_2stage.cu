#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>


#if defined(USE_ROCM)
typedef __hip_bfloat16 nv_bfloat16;
#endif

#include <iostream>
#include <array>
#include <limits>
#include <map>
#include <unordered_map>
#include <vector>
#include <cstdlib>
#include <cstring>

namespace vllm {
constexpr int kMaxBlocks = 36;

#ifndef USE_ROCM
const int defaultBlockLimit = 36;
hipPointer_attribute rangeStartAddrAttr = HIP_POINTER_ATTRIBUTE_RANGE_START_ADDR;
#else
const int defaultBlockLimit = 16;
hipPointer_attribute rangeStartAddrAttr = HIP_POINTER_ATTRIBUTE_RANGE_START_ADDR;
#endif

using FlagType = uint32_t;

struct Signal {
  alignas(128) FlagType start[kMaxBlocks][8];
  alignas(128) FlagType end[kMaxBlocks][8];
  alignas(128) FlagType _flag[kMaxBlocks];
};

struct __align__(16) RankData {
  const void* ptrs[8];
};

struct __align__(16) RankSignals {
  Signal* signals[8];
};

template <typename T, int sz>
struct __align__(alignof(T) * sz) array_t {
  T data[sz];
  using type = T;
  static constexpr int size = sz;
};

template <typename T>
struct packed_t {
  using P = array_t<T, 16 / sizeof(T)>;
  using A = array_t<float, 16 / sizeof(T)>;
};

#define DINLINE __device__ __forceinline__

DINLINE float upcast_s(half val) { return __half2float(val); }

template <typename T>
DINLINE T downcast_s(float val);
template <>
DINLINE half downcast_s(float val) {
  return __float2half(val);
}

DINLINE half& assign_add(half& a, half b) {
  a = __hadd(a, b);
  return a;
}
DINLINE float& assign_add(float& a, float b) { return a += b; }

#if (__CUDA_ARCH__ >= 800 || !defined(__CUDA_ARCH__))
DINLINE float upcast_s(hip_bfloat16 val) { return __bfloat162float(val); }
template <>
DINLINE hip_bfloat16 downcast_s(float val) {
  return __float2bfloat16(val);
}
DINLINE hip_bfloat16& assign_add(hip_bfloat16& a, hip_bfloat16 b) {
  a = __hadd(a, b);
  return a;
}
#endif

template <typename T, int N>
DINLINE array_t<T, N>& packed_assign_add(array_t<T, N>& a, array_t<T, N> b) {
#pragma unroll
  for (int i = 0; i < N; i++) {
    assign_add(a.data[i], b.data[i]);
  }
  return a;
}

template <typename T, int N>
DINLINE array_t<float, N> upcast(array_t<T, N> val) {
  if constexpr (std::is_same<T, float>::value) {
    return val;
  } else {
    array_t<float, N> out;
#pragma unroll
    for (int i = 0; i < N; i++) {
      out.data[i] = upcast_s(val.data[i]);
    }
    return out;
  }
}

template <typename O>
DINLINE O downcast(array_t<float, O::size> val) {
  if constexpr (std::is_same<typename O::type, float>::value) {
    return val;
  } else {
    O out;
#pragma unroll
    for (int i = 0; i < O::size; i++) {
      out.data[i] = downcast_s<typename O::type>(val.data[i]);
    }
    return out;
  }
}

#if !defined(USE_ROCM)

static DINLINE void st_flag_release(FlagType* flag_addr, FlagType flag) {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
  asm volatile("st.release.sys.global.u32 [%1], %0;" : : "r"(flag), "l"(flag_addr));
#else
  asm volatile("membar.sys; st.volatile.global.u32 [%1], %0;" : : "r"(flag), "l"(flag_addr));
#endif
}

static DINLINE FlagType ld_flag_acquire(FlagType* flag_addr) {
  FlagType flag;
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 700
  asm volatile("ld.acquire.sys.global.u32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#else
  asm volatile("ld.volatile.global.u32 %0, [%1]; membar.gl;" : "=r"(flag) : "l"(flag_addr));
#endif
  return flag;
}

static DINLINE void st_flag_volatile(FlagType* flag_addr, FlagType flag) {
  asm volatile("st.volatile.global.u32 [%1], %0;" : : "r"(flag), "l"(flag_addr));
}

static DINLINE FlagType ld_flag_volatile(FlagType* flag_addr) {
  FlagType flag;
  asm volatile("ld.volatile.global.u32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
  return flag;
}

template <int ngpus>
DINLINE void barrier_at_start(const RankSignals& sg, Signal* self_sg, int rank) {
  uint32_t flag = self_sg->_flag[blockIdx.x] + 1;
  if (threadIdx.x < ngpus) {
    auto peer_counter_ptr = &sg.signals[threadIdx.x]->start[blockIdx.x][rank];
    auto self_counter_ptr = &self_sg->start[blockIdx.x][threadIdx.x];
    st_flag_volatile(peer_counter_ptr, flag);
    while (ld_flag_volatile(self_counter_ptr) != flag);
  }
  __syncthreads();
  if (threadIdx.x == 0) self_sg->_flag[blockIdx.x] = flag;
}

template <int ngpus, bool final_sync = false>
DINLINE void barrier_at_end(const RankSignals& sg, Signal* self_sg, int rank) {
  __syncthreads();
  uint32_t flag = self_sg->_flag[blockIdx.x] + 1;
  if (threadIdx.x < ngpus) {
    auto peer_counter_ptr = &sg.signals[threadIdx.x]->end[blockIdx.x][rank];
    auto self_counter_ptr = &self_sg->end[blockIdx.x][threadIdx.x];
    if constexpr (!final_sync) {
      st_flag_release(peer_counter_ptr, flag);
      while (ld_flag_acquire(self_counter_ptr) != flag);
    } else {
      st_flag_volatile(peer_counter_ptr, flag);
      while (ld_flag_volatile(self_counter_ptr) != flag);
    }
  }
  if constexpr (!final_sync) __syncthreads();
  if (threadIdx.x == 0) self_sg->_flag[blockIdx.x] = flag;
}

#else

template <int ngpus>
DINLINE void barrier_at_start(const RankSignals& sg, Signal* self_sg, int rank) {
  uint32_t flag = self_sg->_flag[blockIdx.x] + 1;
  if (threadIdx.x < ngpus) {
    __scoped_atomic_store_n(&sg.signals[threadIdx.x]->start[blockIdx.x][rank], flag, __ATOMIC_RELAXED, __MEMORY_SCOPE_SYSTEM);
    while (__scoped_atomic_load_n(&self_sg->start[blockIdx.x][threadIdx.x], __ATOMIC_RELAXED, __MEMORY_SCOPE_DEVICE) < flag);
  }
  __syncthreads();
  if (threadIdx.x == 0) self_sg->_flag[blockIdx.x] = flag;
}

template <int ngpus, bool final_sync = false>
DINLINE void barrier_at_end(const RankSignals& sg, Signal* self_sg, int rank) {
  __syncthreads();
  uint32_t flag = self_sg->_flag[blockIdx.x] + 1;
  if (threadIdx.x < ngpus) {
    __scoped_atomic_store_n(&sg.signals[threadIdx.x]->end[blockIdx.x][rank], flag, final_sync ? __ATOMIC_RELAXED : __ATOMIC_RELEASE, __MEMORY_SCOPE_SYSTEM);
    while (__scoped_atomic_load_n(&self_sg->end[blockIdx.x][threadIdx.x], final_sync ? __ATOMIC_RELAXED : __ATOMIC_ACQUIRE, __MEMORY_SCOPE_DEVICE) < flag);
  }
  if constexpr (!final_sync) __syncthreads();
  if (threadIdx.x == 0) self_sg->_flag[blockIdx.x] = flag;
}

#endif

template <typename P, int ngpus, typename A>
DINLINE P packed_reduce(const P* ptrs[], int idx) {
  A tmp = upcast(ptrs[0][idx]);
#pragma unroll
  for (int i = 1; i < ngpus; i++) {
    packed_assign_add(tmp, upcast(ptrs[i][idx]));
  }
  return downcast<P>(tmp);
}

template <typename P>
DINLINE P* get_tmp_buf(Signal* sg) {
  return (P*)(((Signal*)sg) + 1);
}

template <typename T, int ngpus>
__global__ void __launch_bounds__(512, 1)
cross_device_reduce_2stage(RankData* _dp, RankSignals sg, Signal* self_sg, T* __restrict__ result, int rank, int size) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;
  using P = typename packed_t<T>::P;
  using A = typename packed_t<T>::A;
  int part = size / ngpus;
  int start = rank * part;
  int end = rank == ngpus - 1 ? size : start + part;
  int largest_part = part + size % ngpus;
  const P* ptrs[ngpus];
  P* tmps[ngpus];
#pragma unroll
  for (int i = 0; i < ngpus; i++) {
    int target = (rank + i) % ngpus;
    ptrs[i] = (const P*)_dp->ptrs[target];
    tmps[i] = get_tmp_buf<P>(sg.signals[target]);
  }
  auto tmp_out = tmps[0];
  barrier_at_start<ngpus>(sg, self_sg, rank);
  for (int idx = start + tid; idx < end; idx += stride) {
    tmp_out[idx - start] = packed_reduce<P, ngpus, A>(ptrs, idx);
  }
  barrier_at_end<ngpus>(sg, self_sg, rank);
  for (int idx = tid; idx < largest_part; idx += stride) {
#pragma unroll
    for (int i = 0; i < ngpus; i++) {
      int gather_from_rank = ((rank + i) % ngpus);
      if (gather_from_rank == ngpus - 1 || idx < part) {
        int dst_idx = gather_from_rank * part + idx;
        ((P*)result)[dst_idx] = tmps[i][idx];
      }
    }
  }
}

}  // namespace vllm
