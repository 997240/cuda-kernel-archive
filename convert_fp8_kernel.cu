#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cfloat>

#ifdef USE_ROCM
  #include <hip/hip_bf16.h>
typedef __hip_bfloat16 __hip_bfloat16;
#endif

namespace vllm {

template <typename Tout, typename Tin, Fp8KVCacheDataType kv_dt>
__global__ void convert_fp8_kernel(const Tin* __restrict__ src_cache,
                                   Tout* __restrict__ dst_cache,
                                   const float scale,
                                   const int64_t block_stride) {
  const int64_t block_idx = blockIdx.x;
  for (int i = threadIdx.x; i < block_stride; i += blockDim.x) {
    int64_t idx = block_idx * block_stride + i;
    dst_cache[idx] =
        fp8::scaled_convert<Tout, Tin, kv_dt>(src_cache[idx], scale);
  }
}

}  // namespace vllm
