#include "hip/hip_runtime.h"
#include <optional>
#include <algorithm>

namespace vllm {

// Implements section 2.2 of https://www.arxiv.org/pdf/2501.01005
// can be used to combine partial attention results (in the split-KV case)
template <typename scalar_t, const uint NUM_THREADS>
__global__ void merge_attn_states_kernel(
    scalar_t* output, float* output_lse, const scalar_t* prefix_output,
    const float* prefix_lse, const scalar_t* suffix_output,
    const float* suffix_lse, const uint num_tokens, const uint num_heads,
    const uint head_size) {
  using pack_128b_t = uint4;
  const uint pack_size = 16 / sizeof(scalar_t);
  const uint threads_per_head = head_size / pack_size;

  const uint global_idx = blockIdx.x * NUM_THREADS + threadIdx.x;
  const uint token_head_threads = num_tokens * num_heads * threads_per_head;

  if (global_idx >= token_head_threads) return;

  // global_idx -> token_idx + head_idx + pack_idx
  const uint token_head_idx = global_idx / threads_per_head;
  const uint pack_idx = global_idx % threads_per_head;

  const uint token_idx = token_head_idx / num_heads;
  const uint head_idx = token_head_idx % num_heads;

  const uint pack_offset = pack_idx * pack_size;  // (0~15)*8, etc.
  const uint head_offset =
      token_idx * num_heads * head_size + head_idx * head_size;
  const scalar_t* prefix_head_ptr = prefix_output + head_offset;
  const scalar_t* suffix_head_ptr = suffix_output + head_offset;
  scalar_t* output_head_ptr = output + head_offset;

  float p_lse = prefix_lse[head_idx * num_tokens + token_idx];
  float s_lse = suffix_lse[head_idx * num_tokens + token_idx];
  p_lse = std::isinf(p_lse) ? -std::numeric_limits<float>::infinity() : p_lse;
  s_lse = std::isinf(s_lse) ? -std::numeric_limits<float>::infinity() : s_lse;

  const float max_lse = fmaxf(p_lse, s_lse);
  p_lse = p_lse - max_lse;
  s_lse = s_lse - max_lse;
  const float p_se = expf(p_lse);
  const float s_se = expf(s_lse);
  const float out_se = p_se + s_se;
  const float p_scale = p_se / out_se;
  const float s_scale = s_se / out_se;

  if (pack_offset < head_size) {
    // Pack 128b load
    pack_128b_t p_out_pack = reinterpret_cast<const pack_128b_t*>(
        prefix_head_ptr)[pack_offset / pack_size];
    pack_128b_t s_out_pack = reinterpret_cast<const pack_128b_t*>(
        suffix_head_ptr)[pack_offset / pack_size];
    pack_128b_t o_out_pack;

#pragma unroll
    for (uint i = 0; i < pack_size; ++i) {
      // Always use float for FMA to keep high precision.
      // half(uint16_t), bfloat16, float -> float.
      const float p_out_f =
          vllm::to_float(reinterpret_cast<const scalar_t*>(&p_out_pack)[i]);
      const float s_out_f =
          vllm::to_float(reinterpret_cast<const scalar_t*>(&s_out_pack)[i]);
      // fma: a * b + c = p_out_f * p_scale + (s_out_f * s_scale)
      const float o_out_f = p_out_f * p_scale + (s_out_f * s_scale);
      // float -> half(uint16_t), bfloat16, float.
      vllm::from_float(reinterpret_cast<scalar_t*>(&o_out_pack)[i], o_out_f);
    }

    // Pack 128b storage
    reinterpret_cast<pack_128b_t*>(output_head_ptr)[pack_offset / pack_size] =
        o_out_pack;
  }
  // We only need to write to output_lse once per head.
  if (output_lse != nullptr && pack_idx == 0) {
    float out_lse = logf(out_se) + max_lse;
    output_lse[head_idx * num_tokens + token_idx] = out_lse;
  }
}

}  // namespace vllm
