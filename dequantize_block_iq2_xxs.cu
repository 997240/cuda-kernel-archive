#include "hip/hip_runtime.h"
template<typename dst_t>
static __global__ void dequantize_block_iq2_xxs(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const auto i   = blockIdx.x;
    const block_iq2_xxs * x = (const block_iq2_xxs  *) vx;

    const auto tid = threadIdx.x;
    const int il = tid/8;
    const int ib = tid%8;
    dst_t * y = yy + i*QK_K + 32*ib + 8*il;
    const uint16_t * q2 = x[i].qs + 4*ib;
    const uint8_t  * aux8 = (const uint8_t *)q2;
    const uint8_t  * grid = (const uint8_t *)(iq2xxs_grid + aux8[il]);
    const uint32_t aux32 = q2[2] | (q2[3] << 16);
    const float d = __half2float(x[i].d) * (0.5f + (aux32 >> 28)) * 0.25f;
    const uint8_t signs = ksigns_iq2xs[(aux32 >> 7*il) & 127];
    for (int j = 0; j < 8; ++j) y[j] = d * grid[j] * (signs & kmask_iq2xs[j] ? -1.f : 1.f);
}