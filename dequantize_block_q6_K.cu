#include "hip/hip_runtime.h"
template<typename dst_t>
static __global__ void dequantize_block_q6_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q6_K * x = (const block_q6_K *) vx;

    const auto i = blockIdx.x;

    const auto tid = threadIdx.x;
    const int ip  = tid/32;
    const int il  = tid - 32*ip;
    const int is  = 8*ip + il/16;

    dst_t * y = yy + i*QK_K + 128*ip + il;

    const half d = x[i].d;

    const uint8_t * ql = x[i].ql + 64*ip + il;
    const uint8_t   qh = x[i].qh[32*ip + il];
    const int8_t  * sc = x[i].scales + is;

    y[ 0] = convert_from_half<dst_t>(__hmul(d, __int2half_rn(sc[0] * ((int8_t)((ql[ 0] & 0xF) | (((qh >> 0) & 3) << 4)) - 32))));
    y[32] = convert_from_half<dst_t>(__hmul(d, __int2half_rn(sc[2] * ((int8_t)((ql[32] & 0xF) | (((qh >> 2) & 3) << 4)) - 32))));
    y[64] = convert_from_half<dst_t>(__hmul(d, __int2half_rn(sc[4] * ((int8_t)((ql[ 0]  >> 4) | (((qh >> 4) & 3) << 4)) - 32))));
    y[96] = convert_from_half<dst_t>(__hmul(d, __int2half_rn(sc[6] * ((int8_t)((ql[32]  >> 4) | (((qh >> 6) & 3) << 4)) - 32))));
}