#include "hip/hip_runtime.h"
template<typename dst_t>
static __global__ void dequantize_block_iq3_xxs(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const auto i   = blockIdx.x;
    const block_iq3_xxs * x = (const block_iq3_xxs  *) vx;

    const auto tid = threadIdx.x;
    const int il = tid/8;
    const int ib = tid%8;
    dst_t * y = yy + i*QK_K + 32*ib + 8*il;
    const uint8_t  * q3 = x[i].qs + 8*ib;
    const uint16_t * gas = (const uint16_t *)(x[i].qs + QK_K/4) + 2*ib;
    const uint8_t  * grid1 = (const uint8_t *)(iq3xxs_grid + q3[2*il+0]);
    const uint8_t  * grid2 = (const uint8_t *)(iq3xxs_grid + q3[2*il+1]);
    const uint32_t aux32 = gas[0] | (gas[1] << 16);
    const float d = __half2float(x[i].d) * (0.5f + (aux32 >> 28)) * 0.5f;
    const uint8_t signs = ksigns_iq2xs[(aux32 >> 7*il) & 127];
    for (int j = 0; j < 4; ++j) {
        y[j+0] = d * grid1[j] * (signs & kmask_iq2xs[j+0] ? -1.f : 1.f);
        y[j+4] = d * grid2[j] * (signs & kmask_iq2xs[j+4] ? -1.f : 1.f);
    }
}