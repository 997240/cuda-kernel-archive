#include "hip/hip_runtime.h"
template<typename dst_t>
static __global__ void dequantize_block_q3_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const auto i = blockIdx.x;
    const block_q3_K * x = (const block_q3_K *) vx;

    const auto r = threadIdx.x/4;
    const int tid = r/2;
    const int is0 = r%2;
    const int l0 = 16*is0 + 4*(threadIdx.x%4);
    const int n = tid / 4;
    const int j = tid - 4*n;

    uint8_t m = 1 << (4*n + j);
    int is = 8*n + 2*j + is0;
    int shift = 2*j;

    int8_t us = is <  4 ? (x[i].scales[is-0] & 0xF) | (((x[i].scales[is+8] >> 0) & 3) << 4) :
                is <  8 ? (x[i].scales[is-0] & 0xF) | (((x[i].scales[is+4] >> 2) & 3) << 4) :
                is < 12 ? (x[i].scales[is-8] >>  4) | (((x[i].scales[is+0] >> 4) & 3) << 4) :
                          (x[i].scales[is-8] >>  4) | (((x[i].scales[is-4] >> 6) & 3) << 4);
    half d_all = x[i].d;
    half dl = __hmul(d_all,  __int2half_rn(us - 32));

    dst_t * y = yy + i*QK_K + 128*n + 32*j;
    const uint8_t * q = x[i].qs + 32*n;
    const uint8_t * hm = x[i].hmask;

    for (int l = l0; l < l0+4; ++l) {
        y[l] = convert_from_half<dst_t>(__hmul(dl,  __int2half_rn((int8_t)((q[l] >> shift) & 3) - ((hm[l] & m) ? 0 : 4))));
    }
}