#include "hip/hip_runtime.h"
static inline __device__ void get_scale_min_k4(int j, const uint8_t * q, uint8_t & d, uint8_t & m) {
    if (j < 4) {
        d = q[j] & 63; m = q[j + 4] & 63;
    } else {
        d = (q[j+4] & 0xF) | ((q[j-4] >> 6) << 4);
        m = (q[j+4] >>  4) | ((q[j-0] >> 6) << 4);
    }
}

template<typename dst_t>
static __global__ void dequantize_block_q4_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q4_K * x = (const block_q4_K *) vx;

    const auto i = blockIdx.x;

    const auto tid = threadIdx.x;
    const int il  = tid/8;
    const int ir  = tid%8;
    const int is  = 2*il;
    const int n   = 4;

    dst_t * y = yy + i*QK_K + 64*il + n*ir;

    const half dall = __low2half(x[i].dm);
    const half dmin = __high2half(x[i].dm);

    const uint8_t * q = x[i].qs + 32*il + n*ir;

    uint8_t sc, m;
    get_scale_min_k4(is + 0, x[i].scales, sc, m);
    const half d1 = __hmul(dall, __int2half_rn(sc));
    const half m1 = __hmul(dmin,  __int2half_rn(m));
    get_scale_min_k4(is + 1, x[i].scales, sc, m);
    const half d2 = __hmul(dall, __int2half_rn(sc));
    const half m2 = __hmul(dmin, __int2half_rn(m));
    for (int l = 0; l < n; ++l) {
        y[l + 0] = convert_from_half<dst_t>(__hsub(__hmul(d1, __int2half_rn(q[l] & 0xF)), m1));
        y[l +32] = convert_from_half<dst_t>(__hsub(__hmul(d2,  __int2half_rn(q[l] >> 4)), m2));
    }
}