#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cfloat>

#ifdef USE_ROCM
  #include <hip/hip_bf16.h>
typedef __hip_bfloat16 __hip_bfloat16;
#endif

namespace vllm {

// Used to copy/convert one element
template <typename OutT, typename InT, Fp8KVCacheDataType kv_dt>
struct CopyWithScaleOp {
  float scale;

  __device__ __forceinline__ void operator()(OutT& dst, const InT src) const {
    if constexpr (kv_dt == Fp8KVCacheDataType::kAuto) {
      dst = static_cast<OutT>(src);
    } else {
      dst = fp8::scaled_convert<OutT, InT, kv_dt>(src, scale);
    }
  }
};

template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void reshape_and_cache_flash_kernel(
    const scalar_t* __restrict__ key,    // [num_tokens, num_heads, head_size]
    const scalar_t* __restrict__ value,  // [num_tokens, num_heads, head_size]
    cache_t* __restrict__ key_cache,     // NHD or HND, shape see comments below
    cache_t* __restrict__ value_cache,   // same above
    const int64_t* __restrict__ slot_mapping,  // [num_tokens]
    const int64_t block_stride, const int64_t page_stride,
    const int64_t head_stride, const int64_t key_stride,
    const int64_t value_stride, const int num_heads, const int head_size,
    const int block_size, const float* k_scale, const float* v_scale) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];
  if (slot_idx < 0) {
    return;
  }
  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;
  const int n_elems = num_heads * head_size;

  const scalar_t* __restrict__ key_src = key + token_idx * key_stride;
  const scalar_t* __restrict__ value_src = value + token_idx * value_stride;

  cache_t* __restrict__ key_dst =
      key_cache + block_idx * block_stride + block_offset * page_stride;
  cache_t* __restrict__ value_dst =
      value_cache + block_idx * block_stride + block_offset * page_stride;

  const bool is_contiguous_heads = (head_stride == head_size);

  float k_scale_val = (kv_dt == Fp8KVCacheDataType::kAuto) ? 0.f : *k_scale;
  float v_scale_val = (kv_dt == Fp8KVCacheDataType::kAuto) ? 0.f : *v_scale;
  constexpr int VEC_SIZE = (sizeof(scalar_t) == 2) ? 8 : 4;
  CopyWithScaleOp<cache_t, scalar_t, kv_dt> k_op{k_scale_val};
  CopyWithScaleOp<cache_t, scalar_t, kv_dt> v_op{v_scale_val};
  if (is_contiguous_heads) {
    vectorize_with_alignment<VEC_SIZE>(key_src, key_dst, n_elems, threadIdx.x,
                                       blockDim.x, k_op);

    vectorize_with_alignment<VEC_SIZE>(value_src, value_dst, n_elems,
                                       threadIdx.x, blockDim.x, v_op);

  } else {
    const int lane = threadIdx.x & 31;
    const int warp_id = threadIdx.x >> 5;
    const int warps_per_block = blockDim.x >> 5;

    for (int head = warp_id; head < num_heads; head += warps_per_block) {
      const scalar_t* __restrict__ k_src_h = key_src + head * head_size;
      const scalar_t* __restrict__ v_src_h = value_src + head * head_size;

      cache_t* __restrict__ k_dst_h =
          key_dst + static_cast<int64_t>(head) * head_stride;
      cache_t* __restrict__ v_dst_h =
          value_dst + static_cast<int64_t>(head) * head_stride;

      vectorize_with_alignment<VEC_SIZE>(k_src_h, k_dst_h, head_size, lane, 32,
                                         k_op);

      vectorize_with_alignment<VEC_SIZE>(v_src_h, v_dst_h, head_size, lane, 32,
                                         v_op);
    }
  }
}

}  // namespace vllm
