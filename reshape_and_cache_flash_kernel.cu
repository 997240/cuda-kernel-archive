#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAException.h>

#include "cuda_utils.h"
#include "cuda_compat.h"
#include "dispatch_utils.h"
#include "quantization/vectorization_utils.cuh"

#ifdef USE_ROCM
  #include "quantization/w8a8/fp8/amd/quant_utils.cuh"
#else
  #include "quantization/w8a8/fp8/nvidia/quant_utils.cuh"
#endif

#include <algorithm>
#include <cassert>
#include <cfloat>

#ifdef USE_ROCM
  #include <hip/hip_bf16.h>
typedef __hip_bfloat16 __hip_bfloat16;
#endif

namespace vllm {

// Used to copy/convert one element
template <typename OutT, typename InT, Fp8KVCacheDataType kv_dt>
struct CopyWithScaleOp {
  float scale;

  __device__ __forceinline__ void operator()(OutT& dst, const InT src) const {
    if constexpr (kv_dt == Fp8KVCacheDataType::kAuto) {
      dst = static_cast<OutT>(src);
    } else {
      dst = fp8::scaled_convert<OutT, InT, kv_dt>(src, scale);
    }
  }
};

template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void reshape_and_cache_flash_kernel(
    const scalar_t* __restrict__ key,    // [num_tokens, num_heads, head_size]
    const scalar_t* __restrict__ value,  // [num_tokens, num_heads, head_size]
    cache_t* __restrict__ key_cache,     // NHD or HND, shape see comments below
    cache_t* __restrict__ value_cache,   // same above
    const int64_t* __restrict__ slot_mapping,  // [num_tokens]
    const int64_t block_stride, const int64_t page_stride,
    const int64_t head_stride, const int64_t key_stride,
    const int64_t value_stride, const int num_heads, const int head_size,
    const int block_size, const float* k_scale, const float* v_scale) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];
  // NOTE: slot_idx can be -1 if the token is padded
  if (slot_idx < 0) {
    return;
  }
  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;
  const int n_elems = num_heads * head_size;

  // pointers to the beginning of the source row for this token.
  const scalar_t* __restrict__ key_src = key + token_idx * key_stride;
  const scalar_t* __restrict__ value_src = value + token_idx * value_stride;

  // find the start position inside the kv-cache for this token.
  cache_t* __restrict__ key_dst =
      key_cache + block_idx * block_stride + block_offset * page_stride;
  cache_t* __restrict__ value_dst =
      value_cache + block_idx * block_stride + block_offset * page_stride;

  // this is true for the NHD layout where `head_stride == head_size`
  const bool is_contiguous_heads = (head_stride == head_size);

  float k_scale_val = (kv_dt == Fp8KVCacheDataType::kAuto) ? 0.f : *k_scale;
  float v_scale_val = (kv_dt == Fp8KVCacheDataType::kAuto) ? 0.f : *v_scale;
  constexpr int VEC_SIZE = (sizeof(scalar_t) == 2) ? 8 : 4;
  CopyWithScaleOp<cache_t, scalar_t, kv_dt> k_op{k_scale_val};
  CopyWithScaleOp<cache_t, scalar_t, kv_dt> v_op{v_scale_val};
  if (is_contiguous_heads) {
    // NHD layout
    // kv cache: [num_blocks, block_size, num_heads, head_size]
    vectorize_with_alignment<VEC_SIZE>(key_src, key_dst, n_elems, threadIdx.x,
                                       blockDim.x, k_op);

    vectorize_with_alignment<VEC_SIZE>(value_src, value_dst, n_elems,
                                       threadIdx.x, blockDim.x, v_op);

  } else {
    // HND layout: heads are strided, but each head_size segment is contiguous
    // kv cache: [num_blocks, num_heads, block_size, head_size]
    const int lane = threadIdx.x & 31;     // 0..31 within warp
    const int warp_id = threadIdx.x >> 5;  // warp index within block
    const int warps_per_block = blockDim.x >> 5;

    for (int head = warp_id; head < num_heads; head += warps_per_block) {
      const scalar_t* __restrict__ k_src_h = key_src + head * head_size;
      const scalar_t* __restrict__ v_src_h = value_src + head * head_size;

      cache_t* __restrict__ k_dst_h =
          key_dst + static_cast<int64_t>(head) * head_stride;
      cache_t* __restrict__ v_dst_h =
          value_dst + static_cast<int64_t>(head) * head_stride;

      // within each head, let the 32 threads of the warp perform the vector
      // copy
      vectorize_with_alignment<VEC_SIZE>(k_src_h, k_dst_h, head_size, lane, 32,
                                         k_op);

      vectorize_with_alignment<VEC_SIZE>(v_src_h, v_dst_h, head_size, lane, 32,
                                         v_op);
    }
  }
}

}  // namespace vllm