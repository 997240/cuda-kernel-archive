#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hipcub/hipcub.hpp>

#include <ATEN/ATEN.h>
#include <ATen/cuda/Atomic.cuh>

namespace vllm {
namespace moe {

template <typename scalar_t>
__global__ void count_and_sort_expert_tokens_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids, int32_t* __restrict__ cumsum_buffer,
    size_t numel, int32_t num_experts) {
  const size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t stride = blockDim.x * gridDim.x;

  for (size_t i = tid; i < numel; i += stride) {
    int32_t expert_id = topk_ids[i];
    if (expert_id >= num_experts) {
      continue;
    }
    int32_t rank_post_pad = atomicAdd(&cumsum_buffer[expert_id], 1);
    sorted_token_ids[rank_post_pad] = i;
  }
}

}  // namespace moe
}  // namespace vllm
