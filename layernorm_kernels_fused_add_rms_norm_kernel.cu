#include "hip/hip_runtime.h"
#include <type_traits>
#include <cstdint>

namespace vllm {

template <typename scalar_t, int width>
__global__ std::enable_if_t<(width == 0) || !_typeConvert<scalar_t>::exists>
fused_add_rms_norm_kernel(
    scalar_t* __restrict__ input,  // [..., hidden_size]
    const int64_t input_stride,
    scalar_t* __restrict__ residual,      // [..., hidden_size]
    const scalar_t* __restrict__ weight,  // [hidden_size]
    const float epsilon, const int num_tokens, const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    scalar_t z = input[blockIdx.x * input_stride + idx];
    z += residual[blockIdx.x * hidden_size + idx];
    float x = (float)z;
    variance += x * x;
    residual[blockIdx.x * hidden_size + idx] = z;
  }

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStore;
  variance = BlockReduce(reduceStore).Reduce(variance, CubAddOp{}, blockDim.x);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)residual[blockIdx.x * hidden_size + idx];
    input[blockIdx.x * input_stride + idx] =
        ((scalar_t)(x * s_variance)) * weight[idx];
  }
}

}  // namespace vllm
