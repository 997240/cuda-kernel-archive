#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void init_rand(hiprandState_t* state, int size, int nRanks) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size;
       idx += gridDim.x * blockDim.x) {
    for (int i = 0; i < nRanks; i++) {
      hiprand_init(i + 1, idx, 0, &state[idx * nRanks + i]);
    }
  }
}