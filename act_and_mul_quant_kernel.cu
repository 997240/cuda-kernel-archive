#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>
#include <cmath>
#include <c10/util/Float8_e4m3fn.h>
#include <array>
#include <type_traits>
#ifndef USE_ROCM
  #include <hip/hip_bf16.h>
  #include <hip/hip_fp16.h>
  #include <hip/hip_fp8.h>
#else
  #include <hip/hip_bf16.h>
  #include <hip/hip_fp16.h>
  #include <hip/hip_fp8.h>

typedef __hip_bfloat162 __hip_bfloat162;
typedef __hip_bfloat16 __hip_bfloat16;
typedef __hip_bfloat16_raw __hip_bfloat16_raw;
  #if defined(HIP_FP8_TYPE_OCP)
typedef __hip_fp8_e4m3 __hip_fp8_e4m3_fnuz;
typedef __hip_fp8x4_e4m3 __hip_fp8x4_e4m3_fnuz;
  #else
// ROCm 6.2 fallback: only *_fnuz types exist
typedef __hip_fp8_e4m3_fnuz __hip_fp8_e4m3_fnuz;
typedef __hip_fp8x4_e4m3_fnuz __hip_fp8x4_e4m3_fnuz;
  #endif
#endif

namespace vllm {

template <typename T>
__device__ __forceinline__ T silu_kernel(const T& x) {
  return (T)(((float)x) / (1.0f + expf((float)-x)));
}

template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&),
          typename fp8_type>
__global__ void act_and_mul_quant_kernel(
    fp8_type* __restrict__ out,
    const scalar_t* __restrict__ input,
    const float* scale,
    const int d) {
  const int32_t blocks_per_token = gridDim.y;

  const int32_t elems_per_128bit_load = (128 / 8) / sizeof(scalar_t);

  const int32_t tgt_elems_per_block = div_ceil(d, blocks_per_token);
  const int32_t elems_per_block =
      round_to_next_multiple_of(tgt_elems_per_block, elems_per_128bit_load);
  const int32_t block_start = blockIdx.y * elems_per_block;
  int32_t block_end = block_start + elems_per_block;
  block_end = block_end > d ? d : block_end;

  const int64_t token_idx = blockIdx.x;
  const scalar_t* __restrict__ x_ptr = input + token_idx * 2 * d;
  const scalar_t* __restrict__ y_ptr = input + token_idx * 2 * d + d;
  fp8_type* __restrict__ out_ptr = out + token_idx * d;

  const int32_t vec_loop_end =
      round_to_previous_multiple_of(elems_per_128bit_load, block_end);
  const int32_t vec_end_idx = vec_loop_end / elems_per_128bit_load;
  const int32_t vec_start_idx = block_start / elems_per_128bit_load;

  const int4* __restrict__ x_128bit_ptr = reinterpret_cast<const int4*>(x_ptr);
  const int4* __restrict__ y_128bit_ptr = reinterpret_cast<const int4*>(y_ptr);
  int2* __restrict__ out_128bit_ptr = reinterpret_cast<int2*>(out_ptr);

  float inverted_scale = 1 / *scale;
#pragma unroll
  for (int32_t vec_idx = vec_start_idx + threadIdx.x; vec_idx < vec_end_idx;
       vec_idx += blockDim.x) {
    const int4 x_128bit = VLLM_LDG(&x_128bit_ptr[vec_idx]);
    const int4 y_128bit = VLLM_LDG(&y_128bit_ptr[vec_idx]);
    using scalar_128bit_vec_t = std::array<scalar_t, elems_per_128bit_load>;
    using scalar_64bit_vec_t = std::array<fp8_type, elems_per_128bit_load>;

    scalar_64bit_vec_t out_vec;
    const auto x_vec = reinterpret_cast<scalar_128bit_vec_t const&>(x_128bit);
    const auto y_vec = reinterpret_cast<scalar_128bit_vec_t const&>(y_128bit);

#pragma unroll
    for (int i = 0; i < elems_per_128bit_load; i++) {
      out_vec[i] = scaled_fp8_conversion<true, fp8_type>(
          ACT_FN(x_vec[i]) * y_vec[i], inverted_scale);
    }

    out_128bit_ptr[vec_idx] = reinterpret_cast<const int2&>(out_vec);
  }

  if (block_end > vec_loop_end) {
    for (int64_t idx = vec_loop_end + threadIdx.x; idx < block_end;
         idx += blockDim.x) {
      const scalar_t x = VLLM_LDG(&x_ptr[idx]);
      const scalar_t y = VLLM_LDG(&y_ptr[idx]);
      out_ptr[idx] =
          scaled_fp8_conversion<true, fp8_type>(ACT_FN(x) * y, inverted_scale);
    }
  }
}

}  // namespace vllm
