#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cstdint>

#ifndef MARLIN_NAMESPACE_NAME
#define MARLIN_NAMESPACE_NAME marlin_moe_wna16
#endif

__host__ __device__ __forceinline__ int div_ceil(int a, int b) {
  return (a + b - 1) / b;
}

constexpr int default_threads = 256;

namespace MARLIN_NAMESPACE_NAME {

template <int moe_block_size>
__global__ void permute_cols_kernel(
    const int4* __restrict__ a_int4_ptr, const int* __restrict__ perm_int_ptr,
    int4* __restrict__ out_int4_ptr,
    const int32_t* __restrict__ sorted_token_ids_ptr,
    const int32_t* __restrict__ expert_ids_ptr,
    const int32_t* __restrict__ num_tokens_past_padded_ptr, int size_m,
    int size_k, int top_k) {
  int num_tokens_past_padded = num_tokens_past_padded_ptr[0];
  int num_moe_blocks = div_ceil(num_tokens_past_padded, moe_block_size);
  int32_t block_sorted_ids[moe_block_size];
  int block_num_valid_tokens = 0;
  int64_t old_expert_id = 0;
  int64_t expert_id = 0;
  int row_stride = size_k * sizeof(half) / 16;

  auto read_moe_block_data = [&](int block_id) {
    block_num_valid_tokens = moe_block_size;
    int4* tmp_block_sorted_ids = reinterpret_cast<int4*>(block_sorted_ids);
    for (int i = 0; i < moe_block_size / 4; i++) {
      tmp_block_sorted_ids[i] =
          reinterpret_cast<const int4*>(sorted_token_ids_ptr)[block_id * moe_block_size / 4 + i];
    }
    for (int i = 0; i < moe_block_size; i++) {
      if (block_sorted_ids[i] >= size_m * top_k) {
        block_num_valid_tokens = i;
        break;
      }
    }
  };

  auto permute_row = [&](int row) {
    int iters = size_k / default_threads;
    int rest = size_k % default_threads;

    int in_offset = (row / top_k) * row_stride;
    int out_offset = row * row_stride;

    const half* a_row_half =
        reinterpret_cast<const half*>(a_int4_ptr + in_offset);
    half* out_half = reinterpret_cast<half*>(out_int4_ptr + out_offset);

    int base_k = 0;

    for (int i = 0; i < iters; i++) {
      int cur_k = base_k + threadIdx.x;
      int src_pos = perm_int_ptr[cur_k];
      out_half[cur_k] = a_row_half[src_pos];
      base_k += default_threads;
    }

    if (rest) {
      if (threadIdx.x < rest) {
        int cur_k = base_k + threadIdx.x;
        int src_pos = perm_int_ptr[cur_k];
        out_half[cur_k] = a_row_half[src_pos];
      }
    }
  };

  for (int index = blockIdx.x; index < num_moe_blocks; index += gridDim.x) {
    old_expert_id = expert_id;
    int tmp_expert_id = expert_ids_ptr[index];
    if (tmp_expert_id == -1) {
      continue;
    }
    expert_id = tmp_expert_id;
    perm_int_ptr += (expert_id - old_expert_id) * size_k;
    read_moe_block_data(index);

    for (int i = 0; i < block_num_valid_tokens; i++) {
      permute_row(block_sorted_ids[i]);
    }
  }
}

} // namespace MARLIN_NAMESPACE_NAME
