#include "hip/hip_runtime.h"
#ifdef USE_ROCM
    #else
    #endif

#ifndef USE_ROCM
    #include <cub/block/block_load.cuh>
    #include <cub/block/block_store.cuh>
    #include <cub/block/block_scan.cuh>
#else
    #include <hipcub/hipcub.hpp>
    namespace cub = hipcub;
#endif

template<int kNThreads_, int kNItems_, int kNRows_, bool kIsEvenLen_,
         bool kIsVariableB_, bool kIsVariableC_,
         bool kHasZ_, bool kVarlen_, typename input_t_, typename weight_t_, typename state_t_>
struct Selective_Scan_fwd_kernel_traits {
    static_assert(kNItems_ % 4 == 0);
    using input_t = input_t_;
    using weight_t = weight_t_;
    using state_t = state_t_;
    static constexpr int kNThreads = kNThreads_;
    static constexpr int kMinBlocks = kNThreads < 128 ? 5 : 3;
    static constexpr int kNItems = kNItems_;
    static constexpr int kNRows = kNRows_;
    static constexpr int kNBytes = sizeof(input_t);
    static_assert(kNBytes == 2 || kNBytes == 4);
    static constexpr int kNElts = kNBytes == 4 ? 4 : constexpr_min(8, kNItems);
    static_assert(kNItems % kNElts == 0);
    static constexpr int kNLoads = kNItems / kNElts;
    static constexpr bool kIsEvenLen = kVarlen_ ? false : kIsEvenLen_;
    static constexpr bool kIsVariableB = kIsVariableB_;
    static constexpr bool kIsVariableC = kIsVariableC_;
    static constexpr bool kHasZ = kHasZ_;
    static constexpr bool kVarlen = kVarlen_;

    static constexpr bool kDirectIO = kVarlen_ ? false : kIsEvenLen && kNLoads == 1;
    static constexpr int kNLoadsIndex = kNItems / 4;
    using vec_t = typename BytesToType<kNBytes * kNElts>::Type;
    using scan_t = float2;
    using BlockLoadT = hipcub::BlockLoad<input_t, kNThreads, kNItems, hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadVecT = hipcub::BlockLoad<vec_t, kNThreads, kNLoads,
        !kDirectIO ? hipcub::BLOCK_LOAD_WARP_TRANSPOSE : hipcub::BLOCK_LOAD_DIRECT>;
    using BlockLoadWeightT = hipcub::BlockLoad<input_t, kNThreads, kNItems , hipcub::BLOCK_LOAD_WARP_TRANSPOSE>;
    using BlockLoadWeightVecT = hipcub::BlockLoad<vec_t, kNThreads, kNLoads ,
        !kDirectIO ? hipcub::BLOCK_LOAD_WARP_TRANSPOSE  : hipcub::BLOCK_LOAD_DIRECT>;
    using BlockStoreT = hipcub::BlockStore<input_t, kNThreads, kNItems, hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
    using BlockStoreVecT = hipcub::BlockStore<vec_t, kNThreads, kNLoads,
        !kDirectIO ? hipcub::BLOCK_STORE_WARP_TRANSPOSE : hipcub::BLOCK_STORE_DIRECT>;
    using BlockScanT = hipcub::BlockScan<scan_t, kNThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
    static constexpr int kSmemIOSize = custom_max({sizeof(typename BlockLoadT::TempStorage),
                                                 sizeof(typename BlockLoadVecT::TempStorage),
                                                 (int(kIsVariableB) + int(kIsVariableC)) * sizeof(typename BlockLoadWeightT::TempStorage),
                                                 (int(kIsVariableB) + int(kIsVariableC)) * sizeof(typename BlockLoadWeightVecT::TempStorage),
                                                 sizeof(typename BlockStoreT::TempStorage),
                                                 sizeof(typename BlockStoreVecT::TempStorage)});
    static constexpr int kSmemSize = kSmemIOSize + sizeof(typename BlockScanT::TempStorage);
};

template<typename Ktraits>
__global__ __launch_bounds__(Ktraits::kNThreads, Ktraits::kMinBlocks)
void selective_scan_fwd_kernel(SSMParamsBase params) {
    constexpr bool kIsVariableB = Ktraits::kIsVariableB;
    constexpr bool kIsVariableC = Ktraits::kIsVariableC;
    constexpr bool kHasZ = Ktraits::kHasZ;
    constexpr bool kVarlen = Ktraits::kVarlen;
    constexpr int kNThreads = Ktraits::kNThreads;
    constexpr int kNItems = Ktraits::kNItems;
    constexpr int kNRows = Ktraits::kNRows;
    constexpr bool kDirectIO = Ktraits::kDirectIO;
    using input_t = typename Ktraits::input_t;
    using weight_t = typename Ktraits::weight_t;
    using scan_t = typename Ktraits::scan_t;

    extern __shared__ char smem_[];
    auto& smem_load = reinterpret_cast<typename Ktraits::BlockLoadT::TempStorage&>(smem_);
    auto& smem_load_weight = reinterpret_cast<typename Ktraits::BlockLoadWeightT::TempStorage&>(smem_);
    auto& smem_load_weight1 = *reinterpret_cast<typename Ktraits::BlockLoadWeightT::TempStorage*>(smem_ + sizeof(typename Ktraits::BlockLoadWeightT::TempStorage));
    auto& smem_store = reinterpret_cast<typename Ktraits::BlockStoreT::TempStorage&>(smem_);
    auto& smem_scan = *reinterpret_cast<typename Ktraits::BlockScanT::TempStorage*>(smem_ + Ktraits::kSmemIOSize);
    scan_t *smem_running_prefix = reinterpret_cast<scan_t *>(smem_ + Ktraits::kSmemSize);

    const int batch_id = blockIdx.x;
    const int dim_id = blockIdx.y;
    const int group_id = dim_id / (params.dim_ngroups_ratio);
    int seqlen = params.seqlen;
    int sequence_start_index = batch_id;
    if constexpr (kVarlen){
        int *query_start_loc = reinterpret_cast<int *>(params.query_start_loc_ptr);
        sequence_start_index = query_start_loc[batch_id];
        seqlen = query_start_loc[batch_id + 1] - sequence_start_index;
    }
    const bool has_initial_state = params.has_initial_state_ptr == nullptr ? false
        : reinterpret_cast<bool *>(params.has_initial_state_ptr)[batch_id];

    const int* cache_indices = params.cache_indices_ptr == nullptr ? nullptr
        : reinterpret_cast<int *>(params.cache_indices_ptr);
    const int cache_index = cache_indices == nullptr ? batch_id : cache_indices[batch_id];
    if (cache_index == params.pad_slot_id){
        return;
    }
    input_t *u = reinterpret_cast<input_t *>(params.u_ptr) + sequence_start_index * params.u_batch_stride
        + dim_id * kNRows * params.u_d_stride;
    input_t *delta = reinterpret_cast<input_t *>(params.delta_ptr) + sequence_start_index * params.delta_batch_stride
        + dim_id * kNRows * params.delta_d_stride;
    weight_t *A = reinterpret_cast<weight_t *>(params.A_ptr) + dim_id * kNRows * params.A_d_stride;
    weight_t *B = reinterpret_cast<weight_t *>(params.B_ptr) + dim_id * kNRows * params.B_d_stride;
    input_t *Bvar = reinterpret_cast<input_t *>(params.B_ptr) + sequence_start_index * params.B_batch_stride + group_id * params.B_group_stride;
    weight_t *C = reinterpret_cast<weight_t *>(params.C_ptr) + dim_id * kNRows * params.C_d_stride;
    input_t *Cvar = reinterpret_cast<input_t *>(params.C_ptr) + sequence_start_index * params.C_batch_stride + group_id * params.C_group_stride;
    typename Ktraits::state_t *ssm_states = reinterpret_cast<typename Ktraits::state_t *>(params.ssm_states_ptr) + 
    cache_index * params.ssm_states_batch_stride + 
    dim_id * kNRows * params.ssm_states_dim_stride;
    
    float D_val[kNRows] = {0};
    if (params.D_ptr != nullptr) {
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            D_val[r] = reinterpret_cast<float *>(params.D_ptr)[dim_id * kNRows + r];
        }
    }
    float delta_bias[kNRows] = {0};
    if (params.delta_bias_ptr != nullptr) {
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            delta_bias[r] = reinterpret_cast<float *>(params.delta_bias_ptr)[dim_id * kNRows + r];
        }
    }

    constexpr int kChunkSize = kNThreads * kNItems;
    const int n_chunks = (seqlen + 2048 - 1) / 2048;
    for (int chunk = 0; chunk < n_chunks; ++chunk) {
        input_t u_vals[kNRows][kNItems], delta_vals_load[kNRows][kNItems];

        __syncthreads();
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            if constexpr (!kDirectIO) {
                if (r > 0) { __syncthreads(); }
            }
            load_input<Ktraits>(u + r * params.u_d_stride, u_vals[r], smem_load, seqlen - chunk * kChunkSize);
            if constexpr (!kDirectIO) { __syncthreads(); }
            load_input<Ktraits>(delta + r * params.delta_d_stride, delta_vals_load[r], smem_load, seqlen - chunk * kChunkSize);
        }
        u += kChunkSize;
        delta += kChunkSize;
    
        float delta_vals[kNRows][kNItems], delta_u_vals[kNRows][kNItems], out_vals[kNRows][kNItems];
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            #pragma unroll
            for (int i = 0; i < kNItems; ++i) {
                float u_val = float(u_vals[r][i]);
                delta_vals[r][i] = float(delta_vals_load[r][i]) + delta_bias[r];
                if (params.delta_softplus) {
                    delta_vals[r][i] = delta_vals[r][i] <= 20.f ? log1pf(expf(delta_vals[r][i])) : delta_vals[r][i];
                }
                delta_u_vals[r][i] = delta_vals[r][i] * u_val;
                out_vals[r][i] = D_val[r] * u_val;
            }
        }

        __syncthreads();
        for (int state_idx = 0; state_idx < params.dstate; ++state_idx) {
            weight_t A_val[kNRows];
            #pragma unroll
            for (int r = 0; r < kNRows; ++r) {
                A_val[r] = A[state_idx * params.A_dstate_stride + r * params.A_d_stride];
                constexpr float kLog2e = M_LOG2E;
                A_val[r] *= kLog2e;
            }
            weight_t BC_val[kNRows];
            weight_t B_vals[kNItems], C_vals[kNItems];
            if constexpr (kIsVariableB) {
                load_weight<Ktraits>(Bvar + state_idx * params.B_dstate_stride, B_vals,
                    smem_load_weight, (seqlen - chunk * kChunkSize) * (1));
                if constexpr (!kIsVariableC) {
                    #pragma unroll
                    for (int r = 0; r < kNRows; ++r) {
                        BC_val[r] = C[state_idx * params.C_dstate_stride + r * params.C_d_stride];
                    }
                }
            }
            if constexpr (kIsVariableC) {
                auto &smem_load_weight_C = !kIsVariableB ? smem_load_weight : smem_load_weight1;
                load_weight<Ktraits>(Cvar + state_idx * params.C_dstate_stride, C_vals,
                    smem_load_weight_C, (seqlen - chunk * kChunkSize) * (1 ));
                if constexpr (!kIsVariableB) {
                    #pragma unroll
                    for (int r = 0; r < kNRows; ++r) {
                        BC_val[r] = B[state_idx * params.B_dstate_stride + r * params.B_d_stride];
                    }
                }
            }
            if constexpr (!kIsVariableB && !kIsVariableC) {
                #pragma unroll
                for (int r = 0; r < kNRows; ++r) {
                    BC_val[r] = B[state_idx * params.B_dstate_stride + r * params.B_d_stride] * C[state_idx * params.C_dstate_stride + r * params.C_d_stride];
                }
            }

            #pragma unroll
            for (int r = 0; r < kNRows; ++r) {
                if (r > 0) { __syncthreads(); }
                scan_t thread_data[kNItems];
                #pragma unroll
                for (int i = 0; i < kNItems; ++i) {
                    thread_data[i] = make_float2(exp2f(delta_vals[r][i] * A_val[r]),
                                                 !kIsVariableB ? delta_u_vals[r][i] : B_vals[i] * delta_u_vals[r][i]);
                    
                    if (seqlen % (kNItems * kNThreads) != 0) {
                        if (threadIdx.x * kNItems + i >= seqlen - chunk * kChunkSize) {
                            thread_data[i] = make_float2(1.f, 0.f);
                        }
                    }
                }

                scan_t running_prefix = chunk > 0 ? smem_running_prefix[state_idx + r * MAX_DSTATE] : make_float2(1.0, has_initial_state ? float(ssm_states[state_idx * params.ssm_states_dstate_stride]): 0.0);

                SSMScanPrefixCallbackOp<weight_t> prefix_op(running_prefix);
                typename Ktraits::BlockScanT(smem_scan).InclusiveScan(
                    thread_data, thread_data, SSMScanOp<weight_t>(), prefix_op
                );
                if (threadIdx.x == 0) {
                    smem_running_prefix[state_idx] = prefix_op.running_prefix;
                    if (chunk == n_chunks - 1) {
                        ssm_states[state_idx * params.ssm_states_dstate_stride] = typename Ktraits::state_t(prefix_op.running_prefix.y);
                    }
                }
                #pragma unroll
                for (int i = 0; i < kNItems; ++i) {
                    const weight_t C_val = !kIsVariableC
                        ? BC_val[r]
                        : (!kIsVariableB ? BC_val[r] * C_vals[i] : C_vals[i]);
                    out_vals[r][i] += thread_data[i].y * C_val;
                }
            }
        }
        
        input_t *out = reinterpret_cast<input_t *>(params.out_ptr) + sequence_start_index * params.out_batch_stride
            + dim_id * kNRows * params.out_d_stride + chunk * kChunkSize;
        __syncthreads();
        #pragma unroll
        for (int r = 0; r < kNRows; ++r) {
            if constexpr (!kDirectIO) {
                if (r > 0) { __syncthreads(); }
            }
            store_output<Ktraits>(out + r * params.out_d_stride, out_vals[r], smem_store, seqlen - chunk * kChunkSize);
        }

        if constexpr (kHasZ) {
            input_t *z = reinterpret_cast<input_t *>(params.z_ptr) + sequence_start_index * params.z_batch_stride
                + dim_id * kNRows * params.z_d_stride + chunk * kChunkSize;
            input_t *out_z = reinterpret_cast<input_t *>(params.out_z_ptr) + sequence_start_index * params.out_z_batch_stride
                + dim_id * kNRows * params.out_z_d_stride + chunk * kChunkSize;
            #pragma unroll
            for (int r = 0; r < kNRows; ++r) {
                input_t z_vals[kNItems];
                __syncthreads();
                load_input<Ktraits>(z + r * params.z_d_stride, z_vals, smem_load, seqlen - chunk * kChunkSize);
                #pragma unroll
                for (int i = 0; i < kNItems; ++i) {
                    float z_val = z_vals[i];
                    out_vals[r][i] *= z_val / (1 + expf(-z_val));
                }
                __syncthreads();
                store_output<Ktraits>(out_z + r * params.out_z_d_stride, out_vals[r], smem_store, seqlen - chunk * kChunkSize);
            }
        }

        Bvar += kChunkSize * 1;
        Cvar += kChunkSize * 1;
    }
}
