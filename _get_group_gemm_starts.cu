#include "hip/hip_runtime.h"
#include <cutlass/arch/arch.h>
#include <cute/tensor.hpp>
#include <cassert>
#include <cstdint>

using namespace cute;

template <typename ElementAB, typename ElementC, typename ElementSF,
          typename ElementAccumulator, typename LayoutSFA, typename LayoutSFB,
          typename ScaleConfig>
__global__ void __get_group_gemm_starts(
    ElementAB** a_offsets, ElementAB** b_offsets, ElementC** out_offsets,
    ElementSF** a_scales_offsets, ElementSF** b_scales_offsets,
    ElementAccumulator** alpha_offsets, LayoutSFA* layout_sfa_base_as_int,
    LayoutSFB* layout_sfb_base_as_int, ElementAB* a_base_as_int,
    ElementAB* b_base_as_int, ElementC* out_base_as_int,
    ElementSF* a_scales_base_as_int, ElementSF* b_scales_base_as_int,
    ElementAccumulator* alphas_base_as_int, const int32_t* expert_offsets,
    const int32_t* sf_offsets, const int32_t* problem_sizes_as_shapes,
    const int K, const int N) {
  int64_t expert_id = threadIdx.x;
  if (expert_id >= gridDim.x * blockDim.x) {
    return;
  }
  // Originally int32_t but upcasting to int64_t to avoid overflow
  // during offset calculations
  int64_t expert_offset = static_cast<int64_t>(expert_offsets[expert_id]);
  int64_t sf_offset = static_cast<int64_t>(sf_offsets[expert_id]);
  // size for block in block scale.
  int64_t group_size = 16;
  int64_t m = static_cast<int64_t>(problem_sizes_as_shapes[expert_id * 3]);
  int64_t n = static_cast<int64_t>(problem_sizes_as_shapes[expert_id * 3 + 1]);
  int64_t k = static_cast<int64_t>(problem_sizes_as_shapes[expert_id * 3 + 2]);
  assert((m >= 0 && n == N && k == K && k % 2 == 0) &&
         "unexpected problem sizes");

  int64_t half_k = static_cast<int64_t>(k / 2);
  int64_t group_k = static_cast<int64_t>(k / group_size);
  // Shape of A as uint8/byte = [M, K // 2]
  // Shape of B as uint8/byte = [E, N, K // 2]
  a_offsets[expert_id] = a_base_as_int + expert_offset * half_k;

  b_offsets[expert_id] = b_base_as_int + expert_id * n * half_k;
  // Shape of C = [M, N]
  out_offsets[expert_id] = out_base_as_int + expert_offset * n;
  // Shape of a_scale = [sum(sf_sizes), K // group_size]
  a_scales_offsets[expert_id] = a_scales_base_as_int + sf_offset * group_k;

  assert((reinterpret_cast<uintptr_t>(a_scales_offsets[expert_id]) % 128) ==
             0 &&
         "TMA requires 128-byte alignment");

  // Shape of B scale = [E, N, K // group_size]
  b_scales_offsets[expert_id] = b_scales_base_as_int + expert_id * n * group_k;
  assert((reinterpret_cast<uintptr_t>(b_scales_offsets[expert_id]) % 128) ==
             0 &&
         "TMA requires 128-byte alignment");
  // Shape of alpha = [E]
  alpha_offsets[expert_id] = alphas_base_as_int + expert_id;

  LayoutSFA* layout_sfa_ptr = layout_sfa_base_as_int + expert_id;
  LayoutSFB* layout_sfb_ptr = layout_sfb_base_as_int + expert_id;

  *layout_sfa_ptr = ScaleConfig::tile_atom_to_shape_SFA(cute::make_shape(
      static_cast<int>(m), static_cast<int>(n), static_cast<int>(k), 1));
  *layout_sfb_ptr = ScaleConfig::tile_atom_to_shape_SFB(cute::make_shape(
      static_cast<int>(m), static_cast<int>(n), static_cast<int>(k), 1));
}
