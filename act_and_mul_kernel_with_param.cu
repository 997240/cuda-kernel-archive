#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>
#include <cmath>

// CUDA compatibility definitions
#ifndef VLLM_LDG
#define VLLM_LDG(ptr) __ldg(ptr)
#endif

namespace vllm {

template <typename T>
__device__ __forceinline__ T fatrelu_kernel(const T& x, const float threshold) {
  const float f = (float)x;
  return (T)(f > threshold ? f : 0.0f);
}

template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&, const float)>
__global__ void act_and_mul_kernel_with_param(
    scalar_t* __restrict__ out, const scalar_t* __restrict__ input, const int d,
    const float param) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = VLLM_LDG(&input[token_idx * 2 * d + idx]);
    const scalar_t y = VLLM_LDG(&input[token_idx * 2 * d + d + idx]);
    out[token_idx * d + idx] = ACT_FN(x, param) * y;
  }
}

}  // namespace vllm