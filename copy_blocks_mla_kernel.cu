#include <stdint.h>
#include <hip/hip_runtime.h>

template <typename scalar_t>
__global__ void copy_blocks_mla_kernel(
    int64_t* cache_ptrs, const int64_t* __restrict__ block_mapping,
    const int mem_footprint_per_block) {
  const int layer_idx = blockIdx.x;
  const int pair_idx = blockIdx.y;
  scalar_t* cache = reinterpret_cast<scalar_t*>(cache_ptrs[layer_idx]);
  int64_t src_block = block_mapping[2 * pair_idx];
  int64_t dst_block = block_mapping[2 * pair_idx + 1];
  int64_t src_offset = src_block * mem_footprint_per_block;
  int64_t dst_offset = dst_block * mem_footprint_per_block;
  for (int i = threadIdx.x; i < mem_footprint_per_block; i += blockDim.x) {
    cache[dst_offset + i] = cache[src_offset + i];
  }
}
