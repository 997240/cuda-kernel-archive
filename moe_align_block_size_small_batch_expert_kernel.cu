#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

#define CEILDIV(x, y) (((x) + (y) - 1) / (y))

namespace vllm {
namespace moe {

template <typename scalar_t>
__global__ void moe_align_block_size_small_batch_expert_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids, int32_t* __restrict__ expert_ids,
    int32_t* __restrict__ total_tokens_post_pad, int32_t num_experts,
    int32_t block_size, size_t numel, int32_t max_num_tokens_padded) {
  for (size_t it = threadIdx.x; it < max_num_tokens_padded; it += blockDim.x) {
    sorted_token_ids[it] = numel;
  }

  const size_t tid = threadIdx.x;
  const size_t stride = blockDim.x;

  extern __shared__ int32_t shared_mem[];
  int32_t* cumsum = shared_mem;
  int32_t* tokens_cnts = (int32_t*)(shared_mem + num_experts + 1);

  for (int i = 0; i < num_experts; ++i) {
    tokens_cnts[(threadIdx.x + 1) * num_experts + i] = 0;
  }

  for (size_t i = tid; i < numel; i += stride) {
    ++tokens_cnts[(threadIdx.x + 1) * num_experts + topk_ids[i]];
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    tokens_cnts[threadIdx.x] = 0;
    for (int i = 1; i <= blockDim.x; ++i) {
      tokens_cnts[i * num_experts + threadIdx.x] +=
          tokens_cnts[(i - 1) * num_experts + threadIdx.x];
    }
  }

  __syncthreads();

  if (threadIdx.x == 0) {
    cumsum[0] = 0;
    for (int i = 1; i <= num_experts; ++i) {
      cumsum[i] =
          cumsum[i - 1] +
          CEILDIV(tokens_cnts[blockDim.x * num_experts + i - 1], block_size) *
              block_size;
    }
    *total_tokens_post_pad = static_cast<int32_t>(cumsum[num_experts]);
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1];
         i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
  }

  const size_t fill_start_idx = cumsum[num_experts] / block_size + threadIdx.x;
  const size_t expert_ids_size = CEILDIV(max_num_tokens_padded, block_size);
  for (size_t i = fill_start_idx; i < expert_ids_size; i += blockDim.x) {
    expert_ids[i] = 0;
  }

  for (size_t i = tid; i < numel; i += stride) {
    int32_t expert_id = topk_ids[i];
    int32_t rank_post_pad =
        tokens_cnts[threadIdx.x * num_experts + expert_id] + cumsum[expert_id];
    sorted_token_ids[rank_post_pad] = i;
    ++tokens_cnts[threadIdx.x * num_experts + expert_id];
  }
}

}  // namespace moe
}  // namespace vllm
