#include "hip/hip_runtime.h"
#include <torch/all.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <hip/hip_fp16.h>

static constexpr int default_threads = 256;

__global__ void permute_cols_kernel(int4 const* __restrict__ a_int4_ptr,
                                    int const* __restrict__ perm_int_ptr,
                                    int4* __restrict__ out_int4_ptr, int size_m,
                                    int size_k, int block_rows) {
  int start_row = block_rows * blockIdx.x;
  int finish_row = start_row + block_rows;
  if (finish_row > size_m) {
    finish_row = size_m;
  }
  int cur_block_rows = std::max(finish_row - start_row, 0);

  int row_stride = size_k * sizeof(half) / 16;

  auto permute_row = [&](int row) {
    int iters = size_k / default_threads;
    int rest = size_k % default_threads;

    int offset = row * row_stride;

    half const* a_row_half = reinterpret_cast<half const*>(a_int4_ptr + offset);
    half* out_half = reinterpret_cast<half*>(out_int4_ptr + offset);

    int base_k = 0;

    for (int i = 0; i < iters; i++) {
      int cur_k = base_k + threadIdx.x;
      int src_pos = perm_int_ptr[cur_k];

      out_half[cur_k] = a_row_half[src_pos];

      base_k += default_threads;
    }

    if (rest) {
      if (threadIdx.x < rest) {
        int cur_k = base_k + threadIdx.x;
        int src_pos = perm_int_ptr[cur_k];

        out_half[cur_k] = a_row_half[src_pos];
      }
    }
  };

  for (int i = 0; i < cur_block_rows; i++) {
    int cur_row = start_row + i;
    if (cur_row < size_m) {
      permute_row(cur_row);
    }
  }
}