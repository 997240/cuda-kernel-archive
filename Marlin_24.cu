#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <iostream>

template <typename T>
inline std::string str(T x) {
  return std::to_string(x);
}

namespace marlin_24 {

static constexpr int THREADS = 256;
static constexpr int STAGES = 4;

static constexpr int min_thread_n = 128;

static constexpr int tile_size = 16;
static constexpr int max_par = 64;

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800

template <const int num_bits,
          const int threads,
          const int thread_m_blocks,
          const int thread_n_blocks,
          const int thread_k_blocks,
          const int stages,
          const int group_blocks = -1>
__global__ void Marlin_24(
    const int4* __restrict__ A,
    const int4* __restrict__ B,
    const int4* __restrict__ meta,
    int4* __restrict__ C,
    const int4* __restrict__ s,
    int prob_m,
    int prob_n,
    int prob_k,
    int* locks
) {}

#else

template <const int num_bits,
          const int threads,
          const int thread_m_blocks,
          const int thread_n_blocks,
          const int thread_k_blocks,
          const int stages,
          const int group_blocks = -1>
__global__ void Marlin_24(
    const int4* __restrict__ A,
    const int4* __restrict__ B,
    const int4* __restrict__ meta,
    int4* __restrict__ C,
    const int4* __restrict__ s,
    int prob_m,
    int prob_n,
    int prob_k,
    int* locks
) {
  int parallel = 1;
  if (prob_m > 16 * thread_m_blocks) {
    parallel = prob_m / (16 * thread_m_blocks);
    prob_m = 16 * thread_m_blocks;
  }

  int k_tiles = prob_k / 32 / thread_k_blocks;
  int n_tiles = prob_n / 16 / thread_n_blocks;
  int iters = ceildiv(k_tiles * n_tiles * parallel, gridDim.x);

  if (group_blocks != -1)
    iters = (group_blocks / thread_k_blocks) *
            ceildiv(iters, (group_blocks / thread_k_blocks));

  int slice_row = (iters * blockIdx.x) % k_tiles;
  int slice_col_par = (iters * blockIdx.x) / k_tiles;
  int slice_col = slice_col_par;
  int slice_iters;
  int slice_count = 0;
  int slice_idx;

  if (slice_col_par >= n_tiles) {
    A += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_k / 8;
    C += (slice_col_par / n_tiles) * 16 * thread_m_blocks * prob_n / 8;
    locks += (slice_col_par / n_tiles) * n_tiles;
    slice_col = slice_col_par % n_tiles;
  }

  auto init_slice = [&]() {
    slice_iters =
        iters * (blockIdx.x + 1) - (k_tiles * slice_col_par + slice_row);
    if (slice_iters < 0 || slice_col_par >= n_tiles * parallel) slice_iters = 0;
    if (slice_iters == 0) return;
    if (slice_row + slice_iters > k_tiles) slice_iters = k_tiles - slice_row;
    slice_count = 1;
    slice_idx = 0;
    int col_first = iters * ceildiv(k_tiles * slice_col_par, iters);
    if (col_first <= k_tiles * (slice_col_par + 1)) {
      int col_off = col_first - k_tiles * slice_col_par;
      slice_count = ceildiv(k_tiles - col_off, iters);
      if (col_off > 0) slice_count++;
      int delta_first = iters * blockIdx.x - col_first;
      if (delta_first < 0 || (col_off == 0 && delta_first == 0))
        slice_idx = slice_count - 1;
      else {
        slice_idx = slice_count - 1 - delta_first / iters;
        if (col_off > 0) slice_idx--;
      }
    }
    if (slice_col == n_tiles) {
      A += 16 * thread_m_blocks * prob_k / 8;
      C += 16 * thread_m_blocks * prob_n / 8;
      locks += n_tiles;
      slice_col = 0;
    }
  };
  init_slice();

  int a_gl_stride = prob_k / 8;

  constexpr int a_sh_stride = 32 * thread_k_blocks / 8;
  constexpr int a_gl_rd_delta_o = 32 * thread_k_blocks / 8;
  int a_gl_rd_delta_i = a_gl_stride * (threads / a_gl_rd_delta_o);
  constexpr int a_sh_wr_delta = a_sh_stride * (threads / a_gl_rd_delta_o);
  constexpr int a_sh_rd_delta_o = 4 * ((threads / 32) / (thread_n_blocks / 4));
  constexpr int a_sh_rd_delta_i = a_sh_stride * 16;
  constexpr int a_sh_stage = a_sh_stride * (16 * thread_m_blocks);
  constexpr int a_sh_wr_iters = ceildiv(a_sh_stage, a_sh_wr_delta);

  constexpr int pack_factor = 32 / num_bits;

  int b_gl_stride = 16 * prob_n / (pack_factor * 4);
  constexpr int b_sh_stride = ((thread_n_blocks * 16) * 16 / pack_factor) / 4;
  constexpr int b_thread_vecs = num_bits == 4 ? 1 : 2;
  constexpr int b_sh_stride_threads = b_sh_stride / b_thread_vecs;
  int b_gl_rd_delta_o = b_gl_stride * thread_k_blocks;
  int b_gl_rd_delta_i = b_gl_stride * (threads / b_sh_stride_threads);
  constexpr int b_sh_wr_delta = threads * b_thread_vecs;
  constexpr int b_sh_rd_delta = threads * b_thread_vecs;
  constexpr int b_sh_stage = b_sh_stride * thread_k_blocks;
  constexpr int b_sh_wr_iters = b_sh_stage / b_sh_wr_delta;

  int m_gl_stride = 2 * prob_n / 8;
  constexpr int m_sh_stride =
      (16 * thread_n_blocks) / 4;
  int m_gl_rd_delta_o = m_gl_stride * thread_k_blocks;
  int m_gl_rd_delta_i = m_gl_stride * (threads / m_sh_stride);
  constexpr int m_sh_wr_delta = threads / 2;
  constexpr int m_sh_rd_delta = threads / 2;
  constexpr int m_sh_stage = m_sh_stride * thread_k_blocks;
  constexpr int m_sh_iters = ceildiv(m_sh_stage, m_sh_wr_delta);

  int s_gl_stride = prob_n / 8;
  constexpr int s_sh_stride = 16 * thread_n_blocks / 8;
  constexpr int s_sh_stage = s_sh_stride;
  int s_gl_rd_delta = s_gl_stride;

  int a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  a_gl_rd += a_gl_rd_delta_o * slice_row;
  int a_sh_wr = a_sh_stride * (threadIdx.x / a_gl_rd_delta_o) +
                (threadIdx.x % a_gl_rd_delta_o);
  int a_sh_rd =
      a_sh_stride * ((threadIdx.x % 32) % 16) + (threadIdx.x % 32) / 16;
  a_sh_rd += 4 * ((threadIdx.x / 32) / (thread_n_blocks / 4));

  int b_gl_rd = b_gl_stride * (threadIdx.x / b_sh_stride_threads) +
                (threadIdx.x % b_sh_stride_threads) * b_thread_vecs;
  b_gl_rd += b_sh_stride * slice_col;
  b_gl_rd += b_gl_rd_delta_o * slice_row;
  auto b_sh_wr = threadIdx.x * b_thread_vecs;
  auto b_sh_rd = threadIdx.x * b_thread_vecs;

  int m_gl_rd = m_gl_stride * (threadIdx.x / (m_sh_stride)) +
                (threadIdx.x % (m_sh_stride));
  m_gl_rd += (m_sh_stride)*slice_col;
  m_gl_rd += m_gl_rd_delta_o * slice_row;
  auto m_sh_wr = threadIdx.x;
  auto m_sh_rd = threadIdx.x % 16 + (threadIdx.x / 32) * 16;

  int s_gl_rd;
  if constexpr (group_blocks == -1) {
    s_gl_rd = s_sh_stride * slice_col + threadIdx.x;
  } else {
    s_gl_rd = s_gl_stride * ((thread_k_blocks * slice_row) / group_blocks) +
              s_sh_stride * slice_col + threadIdx.x;
  }

  auto s_sh_wr = threadIdx.x;
  int s_sh_rd;
  s_sh_rd = 8 * ((threadIdx.x / 32) % (thread_n_blocks / 4)) +
            (threadIdx.x % 32) / 4;

  bool a_sh_wr_pred[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++) {
    a_sh_wr_pred[i] = a_sh_wr_delta * i + a_sh_wr < a_sh_stride * prob_m;
  }
  bool s_sh_wr_pred = threadIdx.x < s_sh_stride;

  auto transform_a = [&](int i) {
    int row = i / a_gl_rd_delta_o;
    return a_gl_rd_delta_o * row + (i % a_gl_rd_delta_o) ^ row;
  };
  int a_sh_wr_trans[a_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < a_sh_wr_iters; i++)
    a_sh_wr_trans[i] = transform_a(a_sh_wr_delta * i + a_sh_wr);
  int a_sh_rd_trans[2][b_sh_wr_iters][thread_m_blocks];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++) {
  #pragma unroll
    for (int j = 0; j < thread_m_blocks; j++) {
      a_sh_rd_trans[0][i][j] =
          transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd);
      a_sh_rd_trans[1][i][j] =
          transform_a(a_sh_rd_delta_o * i + a_sh_rd_delta_i * j + a_sh_rd + 2);
    }
  }

  const int4* B_ptr[b_sh_wr_iters];
  #pragma unroll
  for (int i = 0; i < b_sh_wr_iters; i++)
    B_ptr[i] = B + b_gl_rd_delta_i * i + b_gl_rd;

  bool m_sh_wr_pred = threadIdx.x < m_sh_wr_delta;
  const int4* meta_ptr[m_sh_iters];
  #pragma unroll
  for (int i = 0; i < m_sh_iters; i++)
    meta_ptr[i] = meta + m_gl_rd_delta_i * i + m_gl_rd;

  extern __shared__ int4 sh[];
  int4* sh_a = sh;
  int4* sh_b = sh_a + (stages * a_sh_stage);
  int4* sh_s = sh_b + (stages * b_sh_stage);
  int4* sh_m = sh_s + (stages * s_sh_stage);
  FragA frag_a[2][thread_m_blocks][2];
  I4 frag_b_quant[2][b_thread_vecs];
  FragM frag_m[2][2];
  FragC frag_c[thread_m_blocks][4][2];
  FragS frag_s[2][4];

  auto zero_accums = [&]() {
  #pragma unroll
    for (int i = 0; i < thread_m_blocks * 4 * 2 * 4; i++)
      reinterpret_cast<float*>(frag_c)[i] = 0;
  };

  auto fetch_to_shared = [&](int pipe, int a_off, bool pred = true) {
    if (pred) {
      int4* sh_a_stage = sh_a + a_sh_stage * pipe;
  #pragma unroll
      for (int i = 0; i < a_sh_wr_iters; i++) {
        cp_async4_pred(
            &sh_a_stage[a_sh_wr_trans[i]],
            &A[a_gl_rd_delta_i * i + a_gl_rd + a_gl_rd_delta_o * a_off],
            a_sh_wr_pred[i]);
      }
      int4* sh_b_stage = sh_b + b_sh_stage * pipe;
  #pragma unroll
      for (int i = 0; i < b_sh_wr_iters; i++) {
  #pragma unroll
        for (int j = 0; j < b_thread_vecs; j++) {
          cp_async4(&sh_b_stage[b_sh_wr_delta * i + b_sh_wr + j], B_ptr[i] + j);
        }
        B_ptr[i] += b_gl_rd_delta_o;
      }
      int4* sh_meta_stage = sh_m + m_sh_stage * pipe;
  #pragma unroll
      for (int i = 0; i < m_sh_iters; i++) {
        if (m_sh_wr_pred)
          cp_async4(&sh_meta_stage[m_sh_wr_delta * i + m_sh_wr], meta_ptr[i]);
        meta_ptr[i] += m_gl_rd_delta_o;
      }
      if constexpr (group_blocks != -1) {
        static_assert(group_blocks >= thread_k_blocks);
        if (pipe % (group_blocks / thread_k_blocks) == 0) {
          int4* sh_s_stage = sh_s + s_sh_stage * pipe;
          if (s_sh_wr_pred) cp_async4(&sh_s_stage[s_sh_wr], &s[s_gl_rd]);
          s_gl_rd += s_gl_rd_delta;
        }
      }
    }
    cp_async_fence();
  };

  auto wait_for_stage = [&]() {
    cp_async_wait<stages - 2>();
    __syncthreads();
  };

  auto fetch_to_registers = [&](int k, int pipe) {
    if constexpr (group_blocks != -1) {
      static_assert(group_blocks >= thread_k_blocks);
      int4* sh_s_stage =
          sh_s + s_sh_stage * ((group_blocks / thread_k_blocks) *
                               (pipe / (group_blocks / thread_k_blocks)));
      reinterpret_cast<int4*>(&frag_s[k % 2])[0] = sh_s_stage[s_sh_rd];
    }
    int4* sh_a_stage = sh_a + a_sh_stage * pipe;
  #pragma unroll
    for (int i = 0; i < thread_m_blocks; i++) {
      ldsm4(frag_a[k % 2][i][0],
            &sh_a_stage[a_sh_rd_trans[0][k % b_sh_wr_iters][i]]);
      ldsm4(frag_a[k % 2][i][1],
            &sh_a_stage[a_sh_rd_trans[1][k % b_sh_wr_iters][i]]);
    }

    int4* sh_b_stage = sh_b + b_sh_stage * pipe;
  #pragma unroll
    for (int i = 0; i < b_thread_vecs; i++) {
      frag_b_quant[k % 2][i] = *reinterpret_cast<I4*>(
          &sh_b_stage[b_sh_rd_delta * (k % b_sh_wr_iters) + b_sh_rd + i]);
    }

    int4* sh_m_stage = sh_m + m_sh_stage * pipe;
    ldsm4_m(frag_m[k % 2][0],
            &sh_m_stage[m_sh_rd_delta * (k % m_sh_iters) + m_sh_rd]);
  };

  auto matmul = [&](int k) {
  #pragma unroll
    for (int j = 0; j < 4; j++) {
      FragB frag_b0;
      FragB frag_b1;

      if constexpr (num_bits == 4) {
        int b_quant = frag_b_quant[k % 2][0][j];
        int b_quant_shift = b_quant >> 8;

        frag_b0 = dequant_4bit(b_quant);
        frag_b1 = dequant_4bit(b_quant_shift);

      } else {
        int* frag_b_quant_ptr = reinterpret_cast<int*>(frag_b_quant[k % 2]);
        int b_quant_0 = frag_b_quant_ptr[j * 2 + 0];
        int b_quant_1 = frag_b_quant_ptr[j * 2 + 1];

        frag_b0 = dequant_8bit(b_quant_0);
        frag_b1 = dequant_8bit(b_quant_1);
      }

      if constexpr (group_blocks != -1) {
        scale(frag_b0, frag_s[k % 2][j], 0);
      }
      if constexpr (group_blocks != -1) {
        scale(frag_b1, frag_s[k % 2][j], 1);
      }

  #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        mma_sp(frag_b0, frag_b1, frag_a[k % 2][i][0], frag_c[i][j][0],
               frag_m[k % 2][j / 2], j % 2);
      }
    }
  };

  auto thread_block_reduce = [&]() {
    constexpr int red_off = threads / b_sh_stride_threads / 2;
    if (red_off >= 1) {
      auto red_idx = threadIdx.x / b_sh_stride_threads;
      constexpr int red_sh_stride = b_sh_stride_threads * 4 * 2;
      constexpr int red_sh_delta = b_sh_stride_threads;
      int red_sh_rd = red_sh_stride * (threadIdx.x / b_sh_stride_threads) +
                      (threadIdx.x % b_sh_stride_threads);

  #pragma unroll
      for (int m_block = 0; m_block < thread_m_blocks; m_block++) {
  #pragma unroll
        for (int i = red_off; i > 0; i /= 2) {
          if (i <= red_idx && red_idx < 2 * i) {
  #pragma unroll
            for (int j = 0; j < 4 * 2; j++) {
              int red_sh_wr =
                  red_sh_delta * j + (red_sh_rd - red_sh_stride * i);
              if (i < red_off) {
                float* c_rd =
                    reinterpret_cast<float*>(&sh[red_sh_delta * j + red_sh_rd]);
                float* c_wr = reinterpret_cast<float*>(&sh[red_sh_wr]);
  #pragma unroll
                for (int k = 0; k < 4; k++)
                  reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + j][k] +=
                      c_rd[k] + c_wr[k];
              }
              sh[red_sh_wr] =
                  reinterpret_cast<int4*>(&frag_c)[4 * 2 * m_block + j];
            }
          }
          __syncthreads();
        }
        if (red_idx == 0) {
  #pragma unroll
          for (int i = 0; i < 4 * 2; i++) {
            float* c_rd =
                reinterpret_cast<float*>(&sh[red_sh_delta * i + red_sh_rd]);
  #pragma unroll
            for (int j = 0; j < 4; j++)
              reinterpret_cast<FragC*>(frag_c)[4 * 2 * m_block + i][j] +=
                  c_rd[j];
          }
        }
        __syncthreads();
      }
    }
  };

  auto global_reduce = [&](bool first = false, bool last = false) {
    constexpr int active_threads = 32 * thread_n_blocks / 4;
    if (threadIdx.x < active_threads) {
      int c_gl_stride = prob_n / 8;
      int c_gl_wr_delta_o = 2 * 4 * c_gl_stride;
      int c_gl_wr_delta_i =
          c_gl_stride;
      int c_gl_wr = 2 * c_gl_stride * (threadIdx.x % 4) +
                    8 * (threadIdx.x / 32) + (threadIdx.x % 32) / 4;
      c_gl_wr += (2 * thread_n_blocks) * slice_col;
      constexpr int c_sh_wr_delta = active_threads;
      auto c_sh_wr = threadIdx.x;

      int col = 2 * ((threadIdx.x % 32) % 4);

      if (!first) {
  #pragma unroll
        for (int i = 0; i < thread_m_blocks * 4; i++) {
          cp_async4_pred(&sh[c_sh_wr + c_sh_wr_delta * i],
                         &C[c_gl_wr + c_gl_wr_delta_o * (i / 2) +
                            c_gl_wr_delta_i * (i % 2)],
                         i < (thread_m_blocks - 1) * 4 ||
                             8 * (i / 2) + col + (i % 2) < prob_m);
        }
        cp_async_fence();
        cp_async_wait<0>();
      }

  #pragma unroll
      for (int i = 0; i < thread_m_blocks * 4; i++) {
        if (i < (thread_m_blocks - 1) * 4 ||
            8 * (i / 2) + col + (i % 2) < prob_m) {
          if (!first) {
            int4 c_red = sh[c_sh_wr + i * c_sh_wr_delta];
  #pragma unroll
            for (int j2 = 0; j2 < 2; j2++) {
  #pragma unroll
              for (int j1 = 0; j1 < 4; j1++) {
                reinterpret_cast<float*>(
                    &frag_c)[4 * 2 * 4 * (i / 4) + 8 * j1 + 2 * j2 +
                             4 * ((i % 4) / 2) + i % 2] +=
                    __half2float(
                        reinterpret_cast<__half*>(&c_red)[(j2 * 4 + j1)]);
              }
            }
          }
          if (!last) {
            int4 c;
  #pragma unroll
            for (int j2 = 0; j2 < 2; j2++) {
  #pragma unroll
              for (int j1 = 0; j1 < 4; j1++) {
                reinterpret_cast<__half*>(&c)[(j2 * 4 + j1)] =
                    __float2half(reinterpret_cast<float*>(
                        &frag_c)[4 * 2 * 4 * (i / 4) + 8 * j1 + 2 * j2 +
                                 4 * ((i % 4) / 2) + i % 2]);
              }
            }
            C[c_gl_wr + c_gl_wr_delta_o * (i / 2) + c_gl_wr_delta_i * (i % 2)] =
                c;
          }
        }
      }
    }
  };

  auto write_result = [&]() {
    int c_gl_stride = prob_n / 8;

    constexpr int c_sh_stride = 2 * thread_n_blocks;
    constexpr int c_sh_stride_2 = 2 * c_sh_stride + 2;
    constexpr int c_sh_stride_3 = 2 * (2 * thread_n_blocks) + 2;

    int c_gl_wr_delta = c_gl_stride * (threads / (2 * thread_n_blocks));

    int c_gl_wr = c_gl_stride * (threadIdx.x / (2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * thread_n_blocks));
    c_gl_wr += (2 * thread_n_blocks) * slice_col;

    int c_sh_wr = c_sh_stride_2 * ((threadIdx.x % 32) % 4) +
                  ((threadIdx.x % 32) / 4);
    c_sh_wr += 8 * (threadIdx.x / 32);

    constexpr int c_sh_rd_delta =
        c_sh_stride_3 * (threads / (2 * 2 * thread_n_blocks));
    int c_sh_rd = c_sh_stride_3 * (threadIdx.x / (2 * 2 * thread_n_blocks)) +
                  (threadIdx.x % (2 * 2 * thread_n_blocks));

    int c_gl_wr_end = c_gl_stride * prob_m;

    auto write = [&](int idx, float c0, float c1, float c2, float c3, FragS& s0,
                     float c4, float c5, float c6, float c7, FragS& s1) {
      uint2 res[2];
      res[0] = to_half4(c0, c1, c2, c3);
      res[1] = to_half4(c4, c5, c6, c7);
      half2* tmp = (half2*)&res;
      if constexpr (group_blocks == -1 && num_bits == 4) {
        tmp[0] = __hmul2(tmp[0], s0[0]);
        tmp[1] = __hmul2(tmp[1], s0[1]);
        tmp[2] = __hmul2(tmp[2], s1[0]);
        tmp[3] = __hmul2(tmp[3], s1[1]);
      }
      ((int4*)sh)[idx] = *((int4*)&res[0]);
    };

    if (threadIdx.x / 32 < thread_n_blocks / 4) {
  #pragma unroll
      for (int i = 0; i < thread_m_blocks; i++) {
        int wr = c_sh_wr;
        write(wr, frag_c[i][0][0][0], frag_c[i][1][0][0], frag_c[i][2][0][0],
              frag_c[i][3][0][0], frag_s[0][0], frag_c[i][0][0][2],
              frag_c[i][1][0][2], frag_c[i][2][0][2], frag_c[i][3][0][2],
              frag_s[0][2]);
        write(wr + c_sh_stride, frag_c[i][0][0][1], frag_c[i][1][0][1],
              frag_c[i][2][0][1], frag_c[i][3][0][1], frag_s[0][0],
              frag_c[i][0][0][3], frag_c[i][1][0][3], frag_c[i][2][0][3],
              frag_c[i][3][0][3], frag_s[0][2]);
        write(wr + 4 * c_sh_stride_2, frag_c[i][0][1][0], frag_c[i][1][1][0],
              frag_c[i][2][1][0], frag_c[i][3][1][0], frag_s[0][0],
              frag_c[i][0][1][2], frag_c[i][1][1][2], frag_c[i][2][1][2],
              frag_c[i][3][1][2], frag_s[0][2]);
        write(wr + 4 * c_sh_stride_2 + c_sh_stride, frag_c[i][0][1][1],
              frag_c[i][1][1][1], frag_c[i][2][1][1], frag_c[i][3][1][1],
              frag_s[0][0], frag_c[i][0][1][3], frag_c[i][1][1][3],
              frag_c[i][2][1][3], frag_c[i][3][1][3], frag_s[0][2]);

        c_sh_wr += 8 * c_sh_stride_2;
      }
    }
    __syncthreads();

  #pragma unroll
    for (int i = 0;
         i < ceildiv(16 * thread_m_blocks, threads / (2 * thread_n_blocks));
         i++) {
      if (c_gl_wr < c_gl_wr_end) {
        C[c_gl_wr] = sh[c_sh_rd];
        c_gl_wr += c_gl_wr_delta;
        c_sh_rd += c_sh_rd_delta;
      }
    }
  };

  auto start_pipes = [&]() {
  #pragma unroll
    for (int i = 0; i < stages - 1; i++) fetch_to_shared(i, i, i < slice_iters);
    zero_accums();
    wait_for_stage();
    fetch_to_registers(0, 0);
    a_gl_rd += a_gl_rd_delta_o * (stages - 1);
  };
  start_pipes();

  while (slice_iters) {
  #pragma unroll
    for (int pipe = 0; pipe < stages;) {
      fetch_to_shared((pipe + stages - 1) % stages, pipe,
                      slice_iters >= stages);
      matmul(pipe);
      wait_for_stage();

      fetch_to_registers(pipe + 1, (pipe + 1) % stages);

      pipe++;
      slice_iters--;
      if (slice_iters == 0) break;
    }
    a_gl_rd += a_gl_rd_delta_o * stages;

    if (slice_iters == 0) {
      cp_async_wait<0>();
      bool last = slice_idx == slice_count - 1;
      if constexpr (group_blocks == -1) {
        if constexpr (num_bits == 8) {
          if (s_sh_wr_pred) cp_async4(&sh_s[s_sh_wr], &s[s_gl_rd]);
          cp_async_fence();
        } else {
          if (last) {
            if (s_sh_wr_pred) cp_async4(&sh_s[s_sh_wr], &s[s_gl_rd]);
            cp_async_fence();
          }
        }
      }
      thread_block_reduce();

      if constexpr (group_blocks == -1) {
        if constexpr (num_bits == 8) {
          cp_async_wait<0>();
          __syncthreads();
          if (threadIdx.x / 32 < thread_n_blocks / 4) {
            *(float4*)(frag_s) = *(float4*)(&sh_s[s_sh_rd]);
          }
        } else {
          if (last) {
            cp_async_wait<0>();
            __syncthreads();
            if (threadIdx.x / 32 < thread_n_blocks / 4) {
              *(float4*)(frag_s) = *(float4*)(&sh_s[s_sh_rd]);
            }
          }
        }
      }

      if constexpr (group_blocks == -1 && num_bits == 8) {
        if (threadIdx.x / 32 < thread_n_blocks / 4) {
  #pragma unroll
          for (int i = 0; i < thread_m_blocks; i++) {
            scale_floats(&frag_c[i][0][0][0], &frag_c[i][1][0][0],
                         &frag_c[i][2][0][0], &frag_c[i][3][0][0], frag_s[0][0],
                         &frag_c[i][0][0][2], &frag_c[i][1][0][2],
                         &frag_c[i][2][0][2], &frag_c[i][3][0][2],
                         frag_s[0][2]);

            scale_floats(&frag_c[i][0][0][1], &frag_c[i][1][0][1],
                         &frag_c[i][2][0][1], &frag_c[i][3][0][1], frag_s[0][0],
                         &frag_c[i][0][0][3], &frag_c[i][1][0][3],
                         &frag_c[i][2][0][3], &frag_c[i][3][0][3],
                         frag_s[0][2]);

            scale_floats(&frag_c[i][0][1][0], &frag_c[i][1][1][0],
                         &frag_c[i][2][1][0], &frag_c[i][3][1][0], frag_s[0][0],
                         &frag_c[i][0][1][2], &frag_c[i][1][1][2],
                         &frag_c[i][2][1][2], &frag_c[i][3][1][2],
                         frag_s[0][2]);

            scale_floats(&frag_c[i][0][1][1], &frag_c[i][1][1][1],
                         &frag_c[i][2][1][1], &frag_c[i][3][1][1], frag_s[0][0],
                         &frag_c[i][0][1][3], &frag_c[i][1][1][3],
                         &frag_c[i][2][1][3], &frag_c[i][3][1][3],
                         frag_s[0][2]);
          }
        }
      }

      if (slice_count > 1) {
        barrier_acquire(&locks[slice_col], slice_idx);
        global_reduce(slice_idx == 0, last);
        barrier_release(&locks[slice_col], last);
      }
      if (last)
        write_result();

      slice_row = 0;
      slice_col_par++;
      slice_col++;
      init_slice();
      if (slice_iters) {
        a_gl_rd = a_gl_stride * (threadIdx.x / a_gl_rd_delta_o) +
                  (threadIdx.x % a_gl_rd_delta_o);
  #pragma unroll
        for (int i = 0; i < b_sh_wr_iters; i++)
          B_ptr[i] += b_sh_stride - b_gl_rd_delta_o * k_tiles;
  #pragma unroll
        for (int i = 0; i < m_sh_iters; i++)
          meta_ptr[i] += (m_sh_stride)-m_gl_rd_delta_o * k_tiles;
        if (slice_col == 0) {
  #pragma unroll
          for (int i = 0; i < b_sh_wr_iters; i++) B_ptr[i] -= b_gl_stride;
  #pragma unroll
          for (int i = 0; i < m_sh_iters; i++) meta_ptr[i] -= m_gl_stride;
        }
        s_gl_rd = s_sh_stride * slice_col + threadIdx.x;
        start_pipes();
      }
    }
  }
}

#endif

}  // namespace marlin_24
