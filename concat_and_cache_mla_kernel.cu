#include "hip/hip_runtime.h"

#ifdef USE_ROCM
  #include <hip/hip_bf16.h>
typedef __hip_bfloat16 __hip_bfloat16;
#else
#endif

namespace vllm {

template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void concat_and_cache_mla_kernel(
    const scalar_t* __restrict__ kv_c,  // [num_tokens, kv_lora_rank]
    const scalar_t* __restrict__ k_pe,  // [num_tokens, pe_dim]
    cache_t* __restrict__ kv_cache,  // [num_blocks, block_size, (kv_lora_rank
                                     // + pe_dim)]
    const int64_t* __restrict__ slot_mapping,  // [num_tokens]
    const int block_stride,                    //
    const int entry_stride,                    //
    const int kv_c_stride,                     //
    const int k_pe_stride,                     //
    const int kv_lora_rank,                    //
    const int pe_dim,                          //
    const int block_size,                      //
    const float* scale                         //
) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];
  // NOTE: slot_idx can be -1 if the token is padded
  if (slot_idx < 0) {
    return;
  }
  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;

  auto copy = [&](const scalar_t* __restrict__ src, cache_t* __restrict__ dst,
                  int src_stride, int dst_stride, int size, int offset) {
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
      const int64_t src_idx = token_idx * src_stride + i;
      const int64_t dst_idx =
          block_idx * block_stride + block_offset * entry_stride + i + offset;
      if constexpr (kv_dt == Fp8KVCacheDataType::kAuto) {
        dst[dst_idx] = src[src_idx];
      } else {
        dst[dst_idx] =
            fp8::scaled_convert<cache_t, scalar_t, kv_dt>(src[src_idx], *scale);
      }
    }
  };

  copy(kv_c, kv_cache, kv_c_stride, block_stride, kv_lora_rank, 0);
  copy(k_pe, kv_cache, k_pe_stride, block_stride, pe_dim, kv_lora_rank);
}

}  // namespace vllm