#include "hip/hip_runtime.h"
#include <algorithm>

#ifdef USE_ROCM
#include <hip/hip_bf16.h>
typedef __hip_bfloat16 __hip_bfloat16;
#endif

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

namespace vllm {

template <int NUM_WARPS>
inline __device__ float block_sum(float* red_smem, float sum) {
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    sum += VLLM_SHFL_XOR_SYNC(sum, mask);
  }
  if (lane == 0) {
    red_smem[warp] = sum;
  }
  __syncthreads();
  if (lane < NUM_WARPS) {
    sum = red_smem[lane];
  }
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    sum += VLLM_SHFL_XOR_SYNC(sum, mask);
  }
  return VLLM_SHFL_SYNC(sum, 0);
}

template <typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE,
          int NUM_THREADS, vllm::Fp8KVCacheDataType KV_DTYPE,
          bool IS_BLOCK_SPARSE,
          int PARTITION_SIZE = 0>
__device__ void paged_attention_kernel(
    float* __restrict__ exp_sums,
    float* __restrict__ max_logits,
    scalar_t* __restrict__ out,
    const scalar_t* __restrict__ q,
    const cache_t* __restrict__ k_cache,
    const cache_t* __restrict__ v_cache,
    const int num_kv_heads,
    const float scale,
    const int* __restrict__ block_tables,
    const int* __restrict__ seq_lens,
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    const float* k_scale, const float* v_scale, const int tp_rank,
    const int blocksparse_local_blocks, const int blocksparse_vert_stride,
    const int blocksparse_block_size, const int blocksparse_head_sliding_step) {
  const int seq_idx = blockIdx.y;
  const int partition_idx = blockIdx.z;
  const int max_num_partitions = gridDim.z;
  constexpr bool USE_PARTITIONING = PARTITION_SIZE > 0;
  const int seq_len = seq_lens[seq_idx];
  if (USE_PARTITIONING && partition_idx * PARTITION_SIZE >= seq_len) {
    return;
  }
  const int num_seq_blocks = DIVIDE_ROUND_UP(seq_len, BLOCK_SIZE);
  const int num_blocks_per_partition =
      USE_PARTITIONING ? PARTITION_SIZE / BLOCK_SIZE : num_seq_blocks;
  const int start_block_idx =
      USE_PARTITIONING ? partition_idx * num_blocks_per_partition : 0;
  const int end_block_idx =
      MIN(start_block_idx + num_blocks_per_partition, num_seq_blocks);
  const int num_blocks = end_block_idx - start_block_idx;
  const int start_token_idx = start_block_idx * BLOCK_SIZE;
  const int end_token_idx =
      MIN(start_token_idx + num_blocks * BLOCK_SIZE, seq_len);
  const int num_tokens = end_token_idx - start_token_idx;
  constexpr int THREAD_GROUP_SIZE = MAX(WARP_SIZE / BLOCK_SIZE, 1);
  constexpr int NUM_THREAD_GROUPS =
      NUM_THREADS / THREAD_GROUP_SIZE;
  assert(NUM_THREADS % THREAD_GROUP_SIZE == 0);
  constexpr int NUM_TOKENS_PER_THREAD_GROUP =
      DIVIDE_ROUND_UP(BLOCK_SIZE, WARP_SIZE);
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int thread_idx = threadIdx.x;
  const int warp_idx = thread_idx / WARP_SIZE;
  const int lane = thread_idx % WARP_SIZE;
  const int head_idx = blockIdx.x;
  const int num_heads = gridDim.x;
  const int num_queries_per_kv = num_heads / num_kv_heads;
  const int kv_head_idx = head_idx / num_queries_per_kv;
  const float alibi_slope =
      alibi_slopes == nullptr ? 0.f : alibi_slopes[head_idx];
  constexpr int VEC_SIZE = MAX(16 / (THREAD_GROUP_SIZE * sizeof(scalar_t)), 1);
  using K_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  using Q_vec = typename Vec<scalar_t, VEC_SIZE>::Type;
  using Quant_vec = typename Vec<cache_t, VEC_SIZE>::Type;
  constexpr int NUM_ELEMS_PER_THREAD = HEAD_SIZE / THREAD_GROUP_SIZE;
  constexpr int NUM_VECS_PER_THREAD = NUM_ELEMS_PER_THREAD / VEC_SIZE;
  const int thread_group_idx = thread_idx / THREAD_GROUP_SIZE;
  const int thread_group_offset = thread_idx % THREAD_GROUP_SIZE;
  const scalar_t* q_ptr = q + seq_idx * q_stride + head_idx * HEAD_SIZE;
  __shared__ Q_vec q_vecs[THREAD_GROUP_SIZE][NUM_VECS_PER_THREAD];
#pragma unroll
  for (int i = thread_group_idx; i < NUM_VECS_PER_THREAD;
       i += NUM_THREAD_GROUPS) {
    const int vec_idx = thread_group_offset + i * THREAD_GROUP_SIZE;
    q_vecs[thread_group_offset][i] =
        *reinterpret_cast<const Q_vec*>(q_ptr + vec_idx * VEC_SIZE);
  }
  __syncthreads();
  extern __shared__ char shared_mem[];
  float* logits = reinterpret_cast<float*>(shared_mem);
  __shared__ float red_smem[2 * NUM_WARPS];
  constexpr int x = 16 / sizeof(cache_t);
  float qk_max = -FLT_MAX;
  const int* block_table = block_tables + seq_idx * max_num_blocks_per_seq;
  int bs_block_offset;
  int q_bs_block_id;
  if constexpr (IS_BLOCK_SPARSE) {
    q_bs_block_id = (seq_len - 1) / blocksparse_block_size;
    if (blocksparse_head_sliding_step >= 0)
      bs_block_offset =
          (tp_rank * num_heads + head_idx) * blocksparse_head_sliding_step + 1;
    else
      bs_block_offset = (tp_rank * num_kv_heads + kv_head_idx) *
                            (-blocksparse_head_sliding_step) +
                        1;
  }
  for (int block_idx = start_block_idx + warp_idx; block_idx < end_block_idx;
       block_idx += NUM_WARPS) {
    if constexpr (IS_BLOCK_SPARSE) {
      const int k_bs_block_id = block_idx * BLOCK_SIZE / blocksparse_block_size;
      const bool is_remote =
          ((k_bs_block_id + bs_block_offset) % blocksparse_vert_stride == 0);
      const bool is_local =
          (k_bs_block_id > q_bs_block_id - blocksparse_local_blocks);
      if (!is_remote && !is_local) {
        for (int i = 0; i < NUM_TOKENS_PER_THREAD_GROUP; i++) {
          const int physical_block_offset =
              (thread_group_idx + i * WARP_SIZE) % BLOCK_SIZE;
          const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
          if (thread_group_offset == 0) {
            logits[token_idx - start_token_idx] = -FLT_MAX;
          }
        }
        continue;
      }
    }
    const int64_t physical_block_number =
        static_cast<int64_t>(block_table[block_idx]);
    for (int i = 0; i < NUM_TOKENS_PER_THREAD_GROUP; i++) {
      const int physical_block_offset =
          (thread_group_idx + i * WARP_SIZE) % BLOCK_SIZE;
      const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
      K_vec k_vecs[NUM_VECS_PER_THREAD];
#pragma unroll
      for (int j = 0; j < NUM_VECS_PER_THREAD; j++) {
        const cache_t* k_ptr =
            k_cache + physical_block_number * kv_block_stride +
            kv_head_idx * kv_head_stride + physical_block_offset * x;
        const int vec_idx = thread_group_offset + j * THREAD_GROUP_SIZE;
        const int offset1 = (vec_idx * VEC_SIZE) / x;
        const int offset2 = (vec_idx * VEC_SIZE) % x;
        if constexpr (KV_DTYPE == Fp8KVCacheDataType::kAuto) {
          k_vecs[j] = *reinterpret_cast<const K_vec*>(
              k_ptr + offset1 * BLOCK_SIZE * x + offset2);
        } else {
          Quant_vec k_vec_quant = *reinterpret_cast<const Quant_vec*>(
              k_ptr + offset1 * BLOCK_SIZE * x + offset2);
          k_vecs[j] = fp8::scaled_convert<K_vec, Quant_vec, KV_DTYPE>(
              k_vec_quant, *k_scale);
        }
      }
      float qk = scale * Qk_dot<scalar_t, THREAD_GROUP_SIZE>::dot(
                             q_vecs[thread_group_offset], k_vecs);
      qk += (alibi_slope != 0) ? alibi_slope * (token_idx - seq_len + 1) : 0;
      if (thread_group_offset == 0) {
        const bool mask = token_idx >= seq_len;
        logits[token_idx - start_token_idx] = mask ? 0.f : qk;
        qk_max = mask ? qk_max : fmaxf(qk_max, qk);
      }
    }
  }
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= THREAD_GROUP_SIZE; mask /= 2) {
    qk_max = fmaxf(qk_max, VLLM_SHFL_XOR_SYNC(qk_max, mask));
  }
  if (lane == 0) {
    red_smem[warp_idx] = qk_max;
  }
  __syncthreads();
  qk_max = lane < NUM_WARPS ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    qk_max = fmaxf(qk_max, VLLM_SHFL_XOR_SYNC(qk_max, mask));
  }
  qk_max = VLLM_SHFL_SYNC(qk_max, 0);
  float exp_sum = 0.f;
  for (int i = thread_idx; i < num_tokens; i += NUM_THREADS) {
    float val = __expf(logits[i] - qk_max);
    logits[i] = val;
    exp_sum += val;
  }
  exp_sum = block_sum<NUM_WARPS>(&red_smem[NUM_WARPS], exp_sum);
  const float inv_sum = __fdividef(1.f, exp_sum + 1e-6f);
  for (int i = thread_idx; i < num_tokens; i += NUM_THREADS) {
    logits[i] *= inv_sum;
  }
  __syncthreads();
  if (USE_PARTITIONING && thread_idx == 0) {
    float* max_logits_ptr = max_logits +
                            seq_idx * num_heads * max_num_partitions +
                            head_idx * max_num_partitions + partition_idx;
    *max_logits_ptr = qk_max;
    float* exp_sums_ptr = exp_sums + seq_idx * num_heads * max_num_partitions +
                          head_idx * max_num_partitions + partition_idx;
    *exp_sums_ptr = exp_sum;
  }
  constexpr int V_VEC_SIZE = MIN(16 / sizeof(scalar_t), BLOCK_SIZE);
  using V_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  using L_vec = typename Vec<scalar_t, V_VEC_SIZE>::Type;
  using V_quant_vec = typename Vec<cache_t, V_VEC_SIZE>::Type;
  using Float_L_vec = typename FloatVec<L_vec>::Type;
  constexpr int NUM_V_VECS_PER_ROW = BLOCK_SIZE / V_VEC_SIZE;
  constexpr int NUM_ROWS_PER_ITER = WARP_SIZE / NUM_V_VECS_PER_ROW;
  constexpr int NUM_ROWS_PER_THREAD =
      DIVIDE_ROUND_UP(HEAD_SIZE, NUM_ROWS_PER_ITER);
  float accs[NUM_ROWS_PER_THREAD];
#pragma unroll
  for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
    accs[i] = 0.f;
  }
  scalar_t zero_value;
  zero(zero_value);
  for (int block_idx = start_block_idx + warp_idx; block_idx < end_block_idx;
       block_idx += NUM_WARPS) {
    if constexpr (IS_BLOCK_SPARSE) {
      int v_bs_block_id = block_idx * BLOCK_SIZE / blocksparse_block_size;
      if (!((v_bs_block_id + bs_block_offset) % blocksparse_vert_stride == 0) &&
          !((v_bs_block_id > q_bs_block_id - blocksparse_local_blocks))) {
        continue;
      }
    }
    const int64_t physical_block_number =
        static_cast<int64_t>(block_table[block_idx]);
    const int physical_block_offset = (lane % NUM_V_VECS_PER_ROW) * V_VEC_SIZE;
    const int token_idx = block_idx * BLOCK_SIZE + physical_block_offset;
    L_vec logits_vec;
    from_float(logits_vec, *reinterpret_cast<Float_L_vec*>(logits + token_idx -
                                                           start_token_idx));
    const cache_t* v_ptr = v_cache + physical_block_number * kv_block_stride +
                           kv_head_idx * kv_head_stride;
#pragma unroll
    for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
      const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
      if (row_idx < HEAD_SIZE) {
        const int offset = row_idx * BLOCK_SIZE + physical_block_offset;
        V_vec v_vec;
        if constexpr (KV_DTYPE == Fp8KVCacheDataType::kAuto) {
          v_vec = *reinterpret_cast<const V_vec*>(v_ptr + offset);
        } else {
          V_quant_vec v_quant_vec =
              *reinterpret_cast<const V_quant_vec*>(v_ptr + offset);
          v_vec = fp8::scaled_convert<V_vec, V_quant_vec, KV_DTYPE>(v_quant_vec,
                                                                    *v_scale);
        }
        if (block_idx == num_seq_blocks - 1) {
          scalar_t* v_vec_ptr = reinterpret_cast<scalar_t*>(&v_vec);
#pragma unroll
          for (int j = 0; j < V_VEC_SIZE; j++) {
            v_vec_ptr[j] = token_idx + j < seq_len ? v_vec_ptr[j] : zero_value;
          }
        }
        accs[i] += dot(logits_vec, v_vec);
      }
    }
  }
#pragma unroll
  for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
    float acc = accs[i];
#pragma unroll
    for (int mask = NUM_V_VECS_PER_ROW / 2; mask >= 1; mask /= 2) {
      acc += VLLM_SHFL_XOR_SYNC(acc, mask);
    }
    accs[i] = acc;
  }
  __syncthreads();
  float* out_smem = reinterpret_cast<float*>(shared_mem);
#pragma unroll
  for (int i = NUM_WARPS; i > 1; i /= 2) {
    int mid = i / 2;
    if (warp_idx >= mid && warp_idx < i) {
      float* dst = &out_smem[(warp_idx - mid) * HEAD_SIZE];
#pragma unroll
      for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
        if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
          dst[row_idx] = accs[i];
        }
      }
    }
    __syncthreads();
    if (warp_idx < mid) {
      const float* src = &out_smem[warp_idx * HEAD_SIZE];
#pragma unroll
      for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
        const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
        if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
          accs[i] += src[row_idx];
        }
      }
    }
    __syncthreads();
  }
  if (warp_idx == 0) {
    scalar_t* out_ptr =
        out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
        head_idx * max_num_partitions * HEAD_SIZE + partition_idx * HEAD_SIZE;
#pragma unroll
    for (int i = 0; i < NUM_ROWS_PER_THREAD; i++) {
      const int row_idx = lane / NUM_V_VECS_PER_ROW + i * NUM_ROWS_PER_ITER;
      if (row_idx < HEAD_SIZE && lane % NUM_V_VECS_PER_ROW == 0) {
        from_float(*(out_ptr + row_idx), accs[i]);
      }
    }
  }
}

template <typename scalar_t, typename cache_t, int HEAD_SIZE, int BLOCK_SIZE,
          int NUM_THREADS, vllm::Fp8KVCacheDataType KV_DTYPE,
          bool IS_BLOCK_SPARSE>
__global__ void paged_attention_v1_kernel(
    scalar_t* __restrict__ out,
    const scalar_t* __restrict__ q,
    const cache_t* __restrict__ k_cache,
    const cache_t* __restrict__ v_cache,
    const int num_kv_heads,
    const float scale,
    const int* __restrict__ block_tables,
    const int* __restrict__ seq_lens,
    const int max_num_blocks_per_seq,
    const float* __restrict__ alibi_slopes,
    const int q_stride, const int kv_block_stride, const int kv_head_stride,
    const float* k_scale, const float* v_scale, const int tp_rank,
    const int blocksparse_local_blocks, const int blocksparse_vert_stride,
    const int blocksparse_block_size, const int blocksparse_head_sliding_step) {
  paged_attention_kernel<scalar_t, cache_t, HEAD_SIZE, BLOCK_SIZE, NUM_THREADS,
                         KV_DTYPE, IS_BLOCK_SPARSE>(
      nullptr, nullptr, out, q, k_cache, v_cache, num_kv_heads, scale,
      block_tables, seq_lens, max_num_blocks_per_seq, alibi_slopes, q_stride,
      kv_block_stride, kv_head_stride, k_scale, v_scale, tp_rank,
      blocksparse_local_blocks, blocksparse_vert_stride, blocksparse_block_size,
      blocksparse_head_sliding_step);
}

}  // namespace vllm

#undef MAX
#undef MIN
#undef DIVIDE_ROUND_UP
