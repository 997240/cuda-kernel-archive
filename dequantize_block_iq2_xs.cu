#include "hip/hip_runtime.h"
template<typename dst_t>
static __global__ void dequantize_block_iq2_xs(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const auto i   = blockIdx.x;
    const block_iq2_xs * x = (const block_iq2_xs *) vx;

    const auto tid = threadIdx.x;
    const int il = tid/8;
    const int ib = tid%8;
    dst_t * y = yy + i*QK_K + 32*ib + 8*il;
    const uint16_t * q2 = x[i].qs + 4*ib;
    const uint8_t  * grid = (const uint8_t *)(iq2xs_grid + (q2[il] & 511));
    const float d = __half2float(x[i].d) * (0.5f + ((x[i].scales[ib] >> 4*(il/2)) & 0xf)) * 0.25f;
    const uint8_t signs = ksigns_iq2xs[q2[il] >> 9];
    for (int j = 0; j < 8; ++j) y[j] = d * grid[j] * (signs & kmask_iq2xs[j] ? -1.f : 1.f);

}