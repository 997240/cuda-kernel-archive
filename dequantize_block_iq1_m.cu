#include "hip/hip_runtime.h"
template<typename dst_t>
static __global__ void dequantize_block_iq1_m(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const int64_t i   = blockIdx.x;
    const block_iq1_m * x = (const block_iq1_m  *) vx;

    const int64_t tid = threadIdx.x;
    const int64_t il = tid/8;
    const int64_t ib = tid%8;
    dst_t * y = yy + i*QK_K + 32*ib + 8*il;
    const uint16_t * sc = (const uint16_t *)x[i].scales;
    iq1m_scale_t scale;
    scale.u16 = (sc[0] >> 12) | ((sc[1] >> 8) & 0x00f0) | ((sc[2] >> 4) & 0x0f00) | (sc[3] & 0xf000);
    const int64_t ib16 = 2*ib + il/2;
    const float d = __half2float(scale.f16) * (2*((sc[ib16/4] >> 3*(ib16%4)) & 0x7) + 1);
    const float delta = x[i].qh[2*ib+il/2] & (0x08 << 4*(il%2)) ? -1 - IQ1M_DELTA : -1 + IQ1M_DELTA;
    uint32_t grid32[2]; const int8_t * q = (const int8_t *)grid32;
    grid32[0] = iq1s_grid_gpu[x[i].qs[4*ib+il] | (((x[i].qh[2*ib+il/2] >> 4*(il%2)) & 7) << 8)];
    grid32[1] = (grid32[0] >> 4) & 0x0f0f0f0f;
    grid32[0] &= 0x0f0f0f0f;
    for (int j = 0; j < 8; ++j) {
        y[j] = d * (q[j] + delta);
    }
}