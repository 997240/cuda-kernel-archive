#include "hip/hip_runtime.h"
namespace vllm {

template <typename scalar_t, typename scalar_out_t, bool has_residual = false>
__device__ void rms_norm_dynamic_per_token_quant_vec(
    scalar_out_t* __restrict__ out,
    float* __restrict__ scales,
    scalar_t const* __restrict__ input,
    scalar_t const* __restrict__ weight,
    float const* scale_ub, float const var_epsilon, int32_t const hidden_size,
    scalar_t* __restrict__ residual = nullptr) {
  float rms = 0.0f;
  float token_scale = 0.0f;

  vllm::vectorized::compute_rms<scalar_t, has_residual>(
      &rms, input, hidden_size, var_epsilon, residual);

  vllm::vectorized::compute_dynamic_per_token_scales<scalar_t, scalar_out_t,
                                                     has_residual>(
      &token_scale, scales, input, weight, rms, scale_ub, hidden_size,
      residual);

  if constexpr (std::is_same_v<scalar_out_t, int8_t>) {
    vllm::vectorized::norm_and_quant<scalar_t, scalar_out_t, true,
                                     has_residual>(
        out, input, weight, rms, 1.0f / token_scale, hidden_size, residual);
  } else {
    vllm::vectorized::norm_and_quant<scalar_t, scalar_out_t, false,
                                     has_residual>(
        out, input, weight, rms, token_scale, hidden_size, residual);
  }
}

template <typename scalar_t, typename scalar_out_t, bool has_residual = false>
__global__ void rms_norm_dynamic_per_token_quant_kernel(
    scalar_out_t* __restrict__ out,
    float* __restrict__ scales,
    scalar_t const* __restrict__ input,
    scalar_t const* __restrict__ weight,
    float const* scale_ub, float const var_epsilon, int32_t const hidden_size,
    scalar_t* __restrict__ residual) {
  bool const can_vectorize = hidden_size % 4 == 0;

  if (can_vectorize) {
    return rms_norm_dynamic_per_token_quant_vec<scalar_t, scalar_out_t,
                                                has_residual>(
        out, scales, input, weight, scale_ub, var_epsilon, hidden_size,
        residual);
  }

  float rms = 0.0f;
  float token_scale = 0.0f;

  vllm::compute_rms<scalar_t, has_residual>(&rms, input, hidden_size,
                                            var_epsilon, residual);
  vllm::compute_dynamic_per_token_scales<scalar_t, scalar_out_t, has_residual>(
      &token_scale, scales, input, weight, rms, scale_ub, hidden_size,
      residual);

  if constexpr (std::is_same_v<scalar_out_t, int8_t>) {
    vllm::norm_and_quant<scalar_t, scalar_out_t, true, has_residual>(
        out, input, weight, rms, 1.0f / token_scale, hidden_size, residual);
  } else {
    vllm::norm_and_quant<scalar_t, scalar_out_t, false, has_residual>(
        out, input, weight, rms, token_scale, hidden_size, residual);
  }
}
}  // namespace vllm
