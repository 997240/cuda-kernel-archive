#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

namespace marlin {

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800

__global__ void permute_cols_kernel(int4 const* __restrict__ a_int4_ptr,
                                    int const* __restrict__ perm_int_ptr,
                                    int4* __restrict__ out_int4_ptr, int size_m,
                                    int size_k, int lda, int block_rows) {}

#else

__global__ void permute_cols_kernel(int4 const* __restrict__ a_int4_ptr,
                                    int const* __restrict__ perm_int_ptr,
                                    int4* __restrict__ out_int4_ptr, int size_m,
                                    int size_k, int lda, int block_rows) {
  auto start_row = block_rows * blockIdx.x;
  int finish_row = start_row + block_rows;
  if (finish_row > size_m) {
    finish_row = size_m;
  }
  int cur_block_rows = finish_row - start_row;

  int input_row_stride = lda * sizeof(half) / 16;
  int output_row_stride = size_k * sizeof(half) / 16;

  auto permute_row = [&](int row) {
    int iters = size_k / default_threads;
    int rest = size_k % default_threads;

    int input_offset = row * input_row_stride;
    int output_offset = row * output_row_stride;

    half const* a_row_half =
        reinterpret_cast<half const*>(a_int4_ptr + input_offset);
    half* out_half = reinterpret_cast<half*>(out_int4_ptr + output_offset);

    int base_k = 0;

    for (int i = 0; i < iters; i++) {
      auto cur_k = base_k + threadIdx.x;
      int src_pos = perm_int_ptr[cur_k];

      out_half[cur_k] = a_row_half[src_pos];

      base_k += default_threads;
    }

    if (rest) {
      if (threadIdx.x < rest) {
        auto cur_k = base_k + threadIdx.x;
        int src_pos = perm_int_ptr[cur_k];

        out_half[cur_k] = a_row_half[src_pos];
      }
    }
  };

  for (int i = 0; i < cur_block_rows; i++) {
    int cur_row = start_row + i;
    if (cur_row < size_m) {
      permute_row(cur_row);
    }
  }
}

#endif  // defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 800

}  // namespace marlin
