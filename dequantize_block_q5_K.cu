#include "hip/hip_runtime.h"
static inline __device__ void get_scale_min_k4(int j, const uint8_t * q, uint8_t & d, uint8_t & m) {
    if (j < 4) {
        d = q[j] & 63; m = q[j + 4] & 63;
    } else {
        d = (q[j+4] & 0xF) | ((q[j-4] >> 6) << 4);
        m = (q[j+4] >>  4) | ((q[j-0] >> 6) << 4);
    }
}

template<typename dst_t>
static __global__ void dequantize_block_q5_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q5_K * x = (const block_q5_K *) vx;

    const auto i = blockIdx.x;

    const auto tid = threadIdx.x;
    const int il  = tid/16;
    const int ir  = tid%16;
    const int is  = 2*il;

    dst_t * y = yy + i*QK_K + 64*il + 2*ir;

    const half dall = __low2half(x[i].dm);
    const half dmin = __high2half(x[i].dm);

    const uint8_t * ql = x[i].qs + 32*il + 2*ir;
    const uint8_t * qh = x[i].qh + 2*ir;

    uint8_t sc, m;
    get_scale_min_k4(is + 0, x[i].scales, sc, m);
    const half d1 = __hmul(dall, __int2half_rn(sc)); const half m1 = __hmul(dmin, __int2half_rn(m));
    get_scale_min_k4(is + 1, x[i].scales, sc, m);
    const half d2 = __hmul(dall, __int2half_rn(sc)); const half m2 = __hmul(dmin, __int2half_rn(m));

    uint8_t   hm  = 1 << (2*il);
    y[ 0] = convert_from_half<dst_t>(__hsub(__hmul(d1, __int2half_rn((ql[0] & 0xF) + (qh[0] & hm ? 16 : 0))), m1));
    y[ 1] = convert_from_half<dst_t>(__hsub(__hmul(d1, __int2half_rn((ql[1] & 0xF) + (qh[1] & hm ? 16 : 0))), m1));
    hm <<= 1;
    y[32] = convert_from_half<dst_t>(__hsub(__hmul(d2, __int2half_rn((ql[0] >>  4) + (qh[0] & hm ? 16 : 0))), m2));
    y[33] = convert_from_half<dst_t>(__hsub(__hmul(d2, __int2half_rn((ql[1] >>  4) + (qh[1] & hm ? 16 : 0))), m2));
}