#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <algorithm>

#ifdef USE_ROCM
#include <hip/hip_bf16.h>
typedef __hip_bfloat16 __hip_bfloat16;
#endif

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define DIVIDE_ROUND_UP(a, b) (((a) + (b) - 1) / (b))

namespace vllm {

template <int NUM_WARPS>
inline __device__ float block_sum(float* red_smem, float sum) {
  int warp = threadIdx.x / WARP_SIZE;
  int lane = threadIdx.x % WARP_SIZE;
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    sum += VLLM_SHFL_XOR_SYNC(sum, mask);
  }
  if (lane == 0) {
    red_smem[warp] = sum;
  }
  __syncthreads();
  if (lane < NUM_WARPS) {
    sum = red_smem[lane];
  }
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    sum += VLLM_SHFL_XOR_SYNC(sum, mask);
  }
  return VLLM_SHFL_SYNC(sum, 0);
}

template <typename scalar_t, int HEAD_SIZE, int NUM_THREADS,
          int PARTITION_SIZE>
__global__ void paged_attention_v2_reduce_kernel(
    scalar_t* __restrict__ out,
    const float* __restrict__ exp_sums,
    const float* __restrict__ max_logits,
    const scalar_t* __restrict__ tmp_out,
    const int* __restrict__ seq_lens,
    const int max_num_partitions) {
  const int num_heads = gridDim.x;
  const int head_idx = blockIdx.x;
  const int seq_idx = blockIdx.y;
  const int seq_len = seq_lens[seq_idx];
  const int num_partitions = DIVIDE_ROUND_UP(seq_len, PARTITION_SIZE);
  if (num_partitions == 1) {
    scalar_t* out_ptr =
        out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
    const scalar_t* tmp_out_ptr =
        tmp_out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
        head_idx * max_num_partitions * HEAD_SIZE;
    for (int i = threadIdx.x; i < HEAD_SIZE; i += blockDim.x) {
      out_ptr[i] = tmp_out_ptr[i];
    }
    return;
  }
  constexpr int NUM_WARPS = NUM_THREADS / WARP_SIZE;
  const int warp_idx = threadIdx.x / WARP_SIZE;
  const int lane = threadIdx.x % WARP_SIZE;
  extern __shared__ char shared_mem[];
  __shared__ float red_smem[2 * NUM_WARPS];
  float* shared_max_logits = reinterpret_cast<float*>(shared_mem);
  const float* max_logits_ptr = max_logits +
                                seq_idx * num_heads * max_num_partitions +
                                head_idx * max_num_partitions;
  float max_logit = -FLT_MAX;
  for (int i = threadIdx.x; i < num_partitions; i += blockDim.x) {
    const float l = max_logits_ptr[i];
    shared_max_logits[i] = l;
    max_logit = fmaxf(max_logit, l);
  }
  __syncthreads();
#pragma unroll
  for (int mask = WARP_SIZE / 2; mask >= 1; mask /= 2) {
    max_logit = fmaxf(max_logit, VLLM_SHFL_XOR_SYNC(max_logit, mask));
  }
  if (lane == 0) {
    red_smem[warp_idx] = max_logit;
  }
  __syncthreads();
  max_logit = lane < NUM_WARPS ? red_smem[lane] : -FLT_MAX;
#pragma unroll
  for (int mask = NUM_WARPS / 2; mask >= 1; mask /= 2) {
    max_logit = fmaxf(max_logit, VLLM_SHFL_XOR_SYNC(max_logit, mask));
  }
  max_logit = VLLM_SHFL_SYNC(max_logit, 0);
  float* shared_exp_sums =
      reinterpret_cast<float*>(shared_mem + sizeof(float) * num_partitions);
  const float* exp_sums_ptr = exp_sums +
                              seq_idx * num_heads * max_num_partitions +
                              head_idx * max_num_partitions;
  float global_exp_sum = 0.0f;
  for (int i = threadIdx.x; i < num_partitions; i += blockDim.x) {
    float l = shared_max_logits[i];
    float rescaled_exp_sum = exp_sums_ptr[i] * expf(l - max_logit);
    global_exp_sum += rescaled_exp_sum;
    shared_exp_sums[i] = rescaled_exp_sum;
  }
  __syncthreads();
  global_exp_sum = block_sum<NUM_WARPS>(&red_smem[NUM_WARPS], global_exp_sum);
  const float inv_global_exp_sum = __fdividef(1.0f, global_exp_sum + 1e-6f);
  const scalar_t* tmp_out_ptr =
      tmp_out + seq_idx * num_heads * max_num_partitions * HEAD_SIZE +
      head_idx * max_num_partitions * HEAD_SIZE;
  scalar_t* out_ptr =
      out + seq_idx * num_heads * HEAD_SIZE + head_idx * HEAD_SIZE;
#pragma unroll
  for (int i = threadIdx.x; i < HEAD_SIZE; i += NUM_THREADS) {
    float acc = 0.0f;
    for (int j = 0; j < num_partitions; ++j) {
      acc += to_float(tmp_out_ptr[j * HEAD_SIZE + i]) * shared_exp_sums[j] *
             inv_global_exp_sum;
    }
    from_float(out_ptr[i], acc);
  }
}

}  // namespace vllm

#undef MAX
#undef MIN
#undef DIVIDE_ROUND_UP
