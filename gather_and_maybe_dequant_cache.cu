#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAException.h>

#ifdef USE_ROCM
#else
#endif

#include <algorithm>
#include <cassert>
#include <cfloat>

#ifdef USE_ROCM
  #include <hip/hip_bf16.h>
typedef __hip_bfloat16 __hip_bfloat16;
#endif

namespace vllm {

// grid is launched with dimensions (batch, num_splits)
template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void gather_and_maybe_dequant_cache(
    const cache_t* __restrict__ src_cache,    // [NUM_BLOCKS, BLOCK_SIZE,
                                              // ENTRIES...]
    scalar_t* __restrict__ dst,               // [TOT_TOKENS, ENTRIES...]
    const int32_t* __restrict__ block_table,  // [BATCH, BLOCK_INDICES]
    const int32_t* __restrict__ cu_seq_lens,  // [BATCH+1]
    const int32_t block_size, const int32_t entry_size,
    const int64_t block_table_stride, const int64_t cache_block_stride,
    const int64_t cache_entry_stride, const int64_t dst_entry_stride,
    const float* __restrict__ scale,
    const int32_t* __restrict__ seq_starts) {  // Optional: starting offsets per
                                               // batch

  const int64_t bid = blockIdx.x;  // Batch ID
  const int32_t num_splits = gridDim.y;
  const int32_t split = blockIdx.y;
  const int32_t seq_start = cu_seq_lens[bid];
  const int32_t seq_end = cu_seq_lens[bid + 1];
  const int32_t seq_len = seq_end - seq_start;
  const int32_t tot_blocks = cuda_utils::ceil_div(seq_len, block_size);
  const int32_t split_blocks = cuda_utils::ceil_div(tot_blocks, num_splits);

  const int32_t split_start = split * split_blocks;
  const int32_t split_end = min((split + 1) * split_blocks, tot_blocks);

  const bool is_active_split = (split_start < tot_blocks);
  const bool is_last_split = (split_end == tot_blocks);

  if (!is_active_split) return;

  int32_t full_blocks_end = split_end;
  int32_t partial_block_size = 0;

  // Adjust the pointer for the block_table for this batch.
  // If seq_starts is provided, compute an offset based on (seq_starts[bid] /
  // page_size)
  const int32_t batch_offset = bid * block_table_stride;
  int32_t offset = 0;
  if (seq_starts != nullptr) {
    offset = seq_starts[bid] / block_size;
  }
  const int32_t* batch_block_table = block_table + batch_offset + offset;

  // Adjust dst pointer based on the cumulative sequence lengths.
  dst += seq_start * dst_entry_stride;

  if (is_last_split) {
    partial_block_size = seq_len % block_size;
    if (partial_block_size) full_blocks_end -= 1;
  }

  auto copy_entry = [&](const cache_t* __restrict__ _src,
                        scalar_t* __restrict__ _dst) {
    for (int i = threadIdx.x; i < entry_size; i += blockDim.x) {
      if constexpr (kv_dt == Fp8KVCacheDataType::kAuto) {
        _dst[i] = static_cast<scalar_t>(_src[i]);
      } else {
        _dst[i] =
            fp8::scaled_convert<scalar_t, cache_t, kv_dt>(_src[i], *scale);
      }
    }
  };

  for (int pid = split_start; pid < full_blocks_end; ++pid) {
    auto block_id = batch_block_table[pid];
    auto block_start_ptr = src_cache + block_id * cache_block_stride;
    auto block_dst_ptr = dst + pid * block_size * dst_entry_stride;
    for (int eid = 0; eid < block_size; ++eid) {
      copy_entry(block_start_ptr + eid * cache_entry_stride,
                 block_dst_ptr + eid * dst_entry_stride);
    }
  }

  if (partial_block_size) {
    auto block_id = batch_block_table[full_blocks_end];
    auto block_start_ptr = src_cache + block_id * cache_block_stride;
    auto block_dst_ptr = dst + full_blocks_end * block_size * dst_entry_stride;
    for (int eid = 0; eid < partial_block_size; ++eid) {
      copy_entry(block_start_ptr + eid * cache_entry_stride,
                 block_dst_ptr + eid * dst_entry_stride);
    }
  }
}

}  // namespace vllm