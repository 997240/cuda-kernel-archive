
#include <hip/hip_runtime.h>
#include <cmath>

// CUDA compatibility definitions
#ifndef VLLM_LDG
#define VLLM_LDG(ptr) __ldg(ptr)
#endif

namespace vllm {

// Element-wise activation kernel template.
template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&)>
__global__ void activation_kernel(
    scalar_t* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., d]
    const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = VLLM_LDG(&input[token_idx * d + idx]);
    out[token_idx * d + idx] = ACT_FN(x);
  }
}

template <typename T>
__device__ __forceinline__ T gelu_new_kernel(const T& x) {
  const float x3 = (float)(x * x * x);
  const T t = (T)tanhf((T)(0.79788456f * (float)(x + (T)(0.044715f * x3))));
  return ((T)0.5) * x * (((T)1.0) + t);
}

template <typename T>
__device__ __forceinline__ T gelu_fast_kernel(const T& x) {
  const float f = (float)x;
  const T t =
      (T)tanhf(((T)(f * 0.79788456f)) * (((T)1.0) + (T)(0.044715f * f) * x));
  return ((T)0.5) * x * (((T)1.0) + t);
}

template <typename T>
__device__ __forceinline__ T gelu_quick_kernel(const T& x) {
  // x * sigmoid(1.702 * x)
  return (T)(((float)x) / (1.0f + expf(-1.702f * (float)x)));
}

template <typename T>
__device__ __forceinline__ T silu_kernel(const T& x) {
  // x * sigmoid(x)
  return (T)(((float)x) / (1.0f + expf((float)-x)));
}

template <typename T>
__device__ __forceinline__ T gelu_kernel(const T& x) {
  // Equivalent to PyTorch GELU with 'none' approximation.
  const float f = (float)x;
  constexpr float ALPHA = M_SQRT1_2;
  return (T)(f * 0.5f * (1.0f + ::erf(f * ALPHA)));
}

template <typename T>
__device__ __forceinline__ T gelu_tanh_kernel(const T& x) {
  // Equivalent to PyTorch GELU with 'tanh' approximation.
  const float f = (float)x;
  constexpr float BETA = M_SQRT2 * M_2_SQRTPI * 0.5f;
  constexpr float KAPPA = 0.044715;
  float x_cube = f * f * f;
  float inner = BETA * (f + KAPPA * x_cube);
  return (T)(0.5f * f * (1.0f + ::tanhf(inner)));
}

}  // namespace vllm