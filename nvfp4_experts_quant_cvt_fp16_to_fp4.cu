#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>

namespace vllm {

template <class Type, bool UE8M0_SF = false, bool SMALL_NUM_EXPERTS = false>
__global__ void __launch_bounds__(1024, VLLM_BLOCKS_PER_SM(1024))
    cvt_fp16_to_fp4(int32_t numRows, int32_t numCols, Type const* in,
                    float const* SFScale, uint32_t* out, uint32_t* SFout,
                    uint32_t* input_offset_by_experts,
                    uint32_t* output_scale_offset_by_experts, int n_experts) {
  using PackedVec = PackedVec<Type>;
  static constexpr int CVT_FP4_NUM_THREADS_PER_SF =
      (CVT_FP4_SF_VEC_SIZE / CVT_FP4_ELTS_PER_THREAD);
  static_assert(sizeof(PackedVec) == sizeof(Type) * CVT_FP4_ELTS_PER_THREAD,
                "Vec size is not matched.");
  extern __shared__ uint32_t shared_input_offsets[];

  if constexpr (SMALL_NUM_EXPERTS) {
    for (int i = threadIdx.x; i < n_experts + 1; i += blockDim.x) {
      shared_input_offsets[i] = input_offset_by_experts[i];
    }
  } else {
    for (int i = threadIdx.x * 4; i < n_experts; i += blockDim.x * 4) {
      *reinterpret_cast<int4*>(&shared_input_offsets[i]) =
          *reinterpret_cast<const int4*>(&input_offset_by_experts[i]);
    }
    if (threadIdx.x == 0) {
      shared_input_offsets[n_experts] = input_offset_by_experts[n_experts];
    }
  }

  __syncthreads();

  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int colsPerRow = numCols / CVT_FP4_ELTS_PER_THREAD;

  for (int globalIdx = tid; globalIdx < numRows * colsPerRow;
       globalIdx += gridDim.x * blockDim.x) {
    int rowIdx = globalIdx / colsPerRow;
    int colIdx = globalIdx % colsPerRow;

    int64_t inOffset = rowIdx * colsPerRow + colIdx;
    PackedVec in_vec = reinterpret_cast<PackedVec const*>(in)[inOffset];
    int64_t outOffset = inOffset;
    auto& out_pos = out[outOffset];

    int rowIdx_in_expert = 0;
    int expert_idx = 0;

    int left = 0, right = n_experts - 1;
    while (left <= right) {
      int mid = (left + right) / 2;
      uint32_t mid_offset = shared_input_offsets[mid];
      uint32_t next_offset = shared_input_offsets[mid + 1];

      if (rowIdx >= mid_offset && rowIdx < next_offset) {
        rowIdx_in_expert = rowIdx - mid_offset;
        expert_idx = mid;
        break;
      } else if (rowIdx < mid_offset) {
        right = mid - 1;
      } else {
        left = mid + 1;
      }
    }

    float const SFScaleVal = SFScale == nullptr ? 1.0f : SFScale[expert_idx];

    int factor = CVT_FP4_SF_VEC_SIZE * 4;
    int32_t numCols_padded = (numCols + factor - 1) / factor * factor;
    int numCols_SFout = numCols_padded / CVT_FP4_SF_VEC_SIZE / 4;
    uint32_t* SFout_in_expert =
        SFout + output_scale_offset_by_experts[expert_idx] * numCols_SFout;

    auto sf_out =
        cvt_quant_to_fp4_get_sf_out_offset<uint32_t,
                                           CVT_FP4_NUM_THREADS_PER_SF>(
            rowIdx_in_expert, colIdx, numCols, SFout_in_expert);

    out_pos = cvt_warp_fp16_to_fp4<Type, UE8M0_SF>(in_vec, SFScaleVal, sf_out);
  }
}

}  // namespace vllm
