#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAException.h>

#include "cuda_utils.h"
#include "cuda_compat.h"
#include "dispatch_utils.h"
#include "quantization/vectorization_utils.cuh"

#ifdef USE_ROCM
  #include "quantization/w8a8/fp8/amd/quant_utils.cuh"
#else
  #include "quantization/w8a8/fp8/nvidia/quant_utils.cuh"
#endif

#include <algorithm>
#include <cassert>
#include <cfloat>

#ifdef USE_ROCM
  #include <hip/hip_bf16.h>
typedef __hip_bfloat16 __hip_bfloat16;
#endif

namespace vllm {

template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void concat_and_cache_ds_mla_kernel(
    const scalar_t* __restrict__ kv_c,  // [num_tokens, kv_lora_rank]
    const scalar_t* __restrict__ k_pe,  // [num_tokens, pe_dim]
    cache_t* __restrict__ kv_cache,  // [num_blocks, block_size, (kv_lora_rank
                                     // + pe_dim)]
    const int64_t* __restrict__ slot_mapping,  // [num_tokens]
    const int block_stride,                    //
    const int entry_stride,                    //
    const int kv_c_stride,                     //
    const int k_pe_stride,                     //
    const int kv_lora_rank,                    //
    const int pe_dim,                          //
    const int block_size,                      //
    const float* scale                         //
) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];
  // NOTE: slot_idx can be -1 if the token is padded
  if (slot_idx < 0) {
    return;
  }
  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;
  const int64_t dst_idx_start =
      block_idx * block_stride + block_offset * entry_stride;

  // For the NoPE part, each tile of 128 elements is handled by half of one warp
  // (16 threads). There are 4 total tiles, so 2 warps (64 threads).
  // Lanes 0 and 16 of each warp write the scale values for that warp's tiles.
  // The RoPE part (last 64 elements) is handled by another 1 warp (32 threads).
  // So in total, we use 3 warps (96 threads) per block.

  // Cast kv_cache to 16_bit for RoPE values
  scalar_t* kv_cache_16bit =
      reinterpret_cast<scalar_t*>(&kv_cache[dst_idx_start]);

  // The last warp handles the RoPE part
  if (threadIdx.x >= 64) {
    // Each thread handles two elements of RoPE
    const int8_t pe_idx_start = (threadIdx.x - 64) * 2;
    const int64_t src_idx = token_idx * k_pe_stride + pe_idx_start;
    // Vectorized load of two 16-bit values, performed as one 32-bit load
    const int32_t vals = *reinterpret_cast<const int32_t*>(&k_pe[src_idx]);
    // RoPE values start after the packed 8-bit NoPE values and the
    // 32-bit scales
    const int64_t dst_idx = kv_lora_rank / 2 + 8 + pe_idx_start;
    // Vectorized store of two 16-bit values, performed as one 32-bit store
    *reinterpret_cast<int32_t*>(&kv_cache_16bit[dst_idx]) = vals;
    return;
  }

  // The first two warps handle the NoPE part
  const int8_t warp_idx = threadIdx.x >> 5;
  const int8_t lane_idx = threadIdx.x & 31;
  const int8_t tile_idx = warp_idx * 2 + (lane_idx >> 4);

  // Each thread handles 8 elements of NoPE
  // Load the NoPE elements for this thread into registers
  const int64_t src_idx_start = token_idx * kv_c_stride + (threadIdx.x * 8);
  // Vectorized load of eight 16-bit values, performed as an int4 load
  const int4 vals_i4 = *reinterpret_cast<const int4*>(&kv_c[src_idx_start]);
  const scalar_t* vals = reinterpret_cast<const scalar_t*>(&vals_i4);

  // Max absolute value of this thread's elements
  float max_abs = fmaxf(fmaxf(fmaxf(fabsf(vals[0]), fabsf(vals[1])),
                              fmaxf(fabsf(vals[2]), fabsf(vals[3]))),
                        fmaxf(fmaxf(fabsf(vals[4]), fabsf(vals[5])),
                              fmaxf(fabsf(vals[6]), fabsf(vals[7]))));

  // Warp-level reduction to find the max absolute value in each half-warp
#pragma unroll
  for (int offset = 8; offset > 0; offset /= 2) {
    max_abs = fmaxf(max_abs, VLLM_SHFL_XOR_SYNC_WIDTH(max_abs, offset, 16));
  }

  // Compute the scale for the tile
  float tile_scale = max_abs / 448.f;
  tile_scale = fmaxf(tile_scale, FLT_MIN);

  // The first lane of each half-warp writes the scale to kv_cache
  if ((lane_idx == 0) || (lane_idx == 16)) {
    float* kv_cache_32bit = reinterpret_cast<float*>(&kv_cache[dst_idx_start]);
    const uint64_t dst_idx = kv_lora_rank / 4 + tile_idx;
    kv_cache_32bit[dst_idx] = tile_scale;
  }

  // Now all threads in the block scale and write their elements
  // NoPE data is packed in the first kv_lora_rank/2 bytes (first 256 bytes)
  const int64_t dst_idx_base = dst_idx_start + (threadIdx.x * 8);

  uint8_t result[8];
#pragma unroll
  for (int i = 0; i < 8; i++) {
    result[i] =
        fp8::scaled_convert<uint8_t, scalar_t, Fp8KVCacheDataType::kFp8E4M3>(
            vals[i], tile_scale);
  }

  // Store as aligned 64-bit writes
  *reinterpret_cast<uint64_t*>(&kv_cache[dst_idx_base]) =
      *reinterpret_cast<const uint64_t*>(result);
}

}  // namespace vllm