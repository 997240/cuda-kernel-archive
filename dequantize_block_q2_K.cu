#include "hip/hip_runtime.h"
template<typename dst_t>
static __global__ void dequantize_block_q2_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const auto i   = blockIdx.x;
    const block_q2_K * x = (const block_q2_K *) vx;

    const auto tid = threadIdx.x;
    const int n   = tid/32;
    const int l   = tid - 32*n;
    const int is  = 8*n + l/16;

    const uint8_t q = x[i].qs[32*n + l];
    dst_t * y = yy + i*QK_K + 128*n;

    half dall = __low2half(x[i].dm);
    half dmin = __high2half(x[i].dm);
    y[l+ 0] = convert_from_half<dst_t>(__hsub(__hmul(dall, __int2half_rn((x[i].scales[is+0] & 0xF) * ((q >> 0) & 3))), __hmul(dmin,  __int2half_rn(x[i].scales[is+0] >> 4))));
    y[l+32] = convert_from_half<dst_t>(__hsub(__hmul(dall, __int2half_rn((x[i].scales[is+2] & 0xF) * ((q >> 2) & 3))), __hmul(dmin,  __int2half_rn(x[i].scales[is+2] >> 4))));
    y[l+64] = convert_from_half<dst_t>(__hsub(__hmul(dall, __int2half_rn((x[i].scales[is+4] & 0xF) * ((q >> 4) & 3))), __hmul(dmin,  __int2half_rn(x[i].scales[is+4] >> 4))));
    y[l+96] = convert_from_half<dst_t>(__hsub(__hmul(dall, __int2half_rn((x[i].scales[is+6] & 0xF) * ((q >> 6) & 3))), __hmul(dmin,  __int2half_rn(x[i].scales[is+6] >> 4))));
}