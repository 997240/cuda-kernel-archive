#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

#define CEILDIV(x, y) (((x) + (y) - 1) / (y))
#ifndef WARP_SIZE
#define WARP_SIZE 32
#endif

namespace vllm {
namespace moe {

template <typename scalar_t>
__global__ void moe_align_block_size_kernel(
    const scalar_t* __restrict__ topk_ids,
    int32_t* __restrict__ sorted_token_ids, int32_t* __restrict__ expert_ids,
    int32_t* __restrict__ total_tokens_post_pad, int32_t num_experts,
    int32_t padded_num_experts, int32_t experts_per_warp, int32_t block_size,
    size_t numel, int32_t* __restrict__ cumsum, int32_t max_num_tokens_padded) {
  extern __shared__ int32_t shared_counts[];

  for (size_t it = threadIdx.x; it < max_num_tokens_padded; it += blockDim.x) {
    sorted_token_ids[it] = numel;
  }

  const int warp_id = threadIdx.x / WARP_SIZE;
  const int my_expert_start = warp_id * experts_per_warp;

  for (int i = 0; i < experts_per_warp; ++i) {
    if (my_expert_start + i < padded_num_experts) {
      shared_counts[warp_id * experts_per_warp + i] = 0;
    }
  }

  __syncthreads();

  const size_t tid = threadIdx.x;
  const size_t stride = blockDim.x;

  for (size_t i = tid; i < numel; i += stride) {
    int expert_id = topk_ids[i];
    if (expert_id >= num_experts) {
      continue;
    }
    int warp_idx = expert_id / experts_per_warp;
    int expert_offset = expert_id % experts_per_warp;
    atomicAdd(&shared_counts[warp_idx * experts_per_warp + expert_offset], 1);
  }

  __syncthreads();

  using BlockScan = hipcub::BlockScan<int32_t, 1024>;
  __shared__ typename BlockScan::TempStorage temp_storage;

  int expert_count = 0;
  int expert_id = threadIdx.x;
  if (expert_id < num_experts) {
    int warp_idx = expert_id / experts_per_warp;
    int expert_offset = expert_id % experts_per_warp;
    expert_count = shared_counts[warp_idx * experts_per_warp + expert_offset];
    expert_count = CEILDIV(expert_count, block_size) * block_size;
  }

  int cumsum_val;
  BlockScan(temp_storage).ExclusiveSum(expert_count, cumsum_val);
  if (expert_id <= num_experts) {
    cumsum[expert_id] = cumsum_val;
  }

  if (expert_id == num_experts) {
    *total_tokens_post_pad = cumsum_val;
  }

  __syncthreads();

  if (threadIdx.x < num_experts) {
    for (int i = cumsum[threadIdx.x]; i < cumsum[threadIdx.x + 1];
         i += block_size) {
      expert_ids[i / block_size] = threadIdx.x;
    }
  }

  const size_t fill_start_idx = cumsum[num_experts] / block_size + threadIdx.x;
  const size_t expert_ids_size = CEILDIV(max_num_tokens_padded, block_size);
  for (size_t i = fill_start_idx; i < expert_ids_size; i += blockDim.x) {
    expert_ids[i] = 0;
  }
}

}  // namespace moe
}  // namespace vllm
