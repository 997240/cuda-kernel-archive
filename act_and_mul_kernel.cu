#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <c10/cuda/CUDAGuard.h>
#include <cmath>

// CUDA compatibility definitions
#ifndef VLLM_LDG
#define VLLM_LDG(ptr) __ldg(ptr)
#endif

namespace vllm {

template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&),
          bool act_first>
__device__ __forceinline__ scalar_t compute(const scalar_t& x,
                                            const scalar_t& y) {
  return act_first ? ACT_FN(x) * y : x * ACT_FN(y);
}

// Activation and gating kernel template.
template <typename scalar_t, scalar_t (*ACT_FN)(const scalar_t&),
          bool act_first>
__global__ void act_and_mul_kernel(
    scalar_t* __restrict__ out,          // [..., d]
    const scalar_t* __restrict__ input,  // [..., 2, d]
    const int d) {
  const int64_t token_idx = blockIdx.x;
  for (int64_t idx = threadIdx.x; idx < d; idx += blockDim.x) {
    const scalar_t x = VLLM_LDG(&input[token_idx * 2 * d + idx]);
    const scalar_t y = VLLM_LDG(&input[token_idx * 2 * d + d + idx]);
    out[token_idx * d + idx] = compute<scalar_t, ACT_FN, act_first>(x, y);
  }
}

template <typename T>
__device__ __forceinline__ T silu_kernel(const T& x) {
  // x * sigmoid(x)
  return (T)(((float)x) / (1.0f + expf((float)-x)));
}

template <typename T>
__device__ __forceinline__ T gelu_kernel(const T& x) {
  // Equivalent to PyTorch GELU with 'none' approximation.
  const float f = (float)x;
  constexpr float ALPHA = M_SQRT1_2;
  return (T)(f * 0.5f * (1.0f + ::erf(f * ALPHA)));
}

template <typename T>
__device__ __forceinline__ T gelu_tanh_kernel(const T& x) {
  // Equivalent to PyTorch GELU with 'tanh' approximation.
  const float f = (float)x;
  constexpr float BETA = M_SQRT2 * M_2_SQRTPI * 0.5f;
  constexpr float KAPPA = 0.044715;
  float x_cube = f * f * f;
  float inner = BETA * (f + KAPPA * x_cube);
  return (T)(0.5f * f * (1.0f + ::tanhf(inner)));
}

}  // namespace vllm