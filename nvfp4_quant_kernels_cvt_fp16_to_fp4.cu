#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>

namespace vllm {

template <class Type, bool UE8M0_SF = false>
__global__ void __launch_bounds__(512, VLLM_BLOCKS_PER_SM(512))
    cvt_fp16_to_fp4(int32_t numRows, int32_t numCols, Type const* in,
                    float const* SFScale, uint32_t* out, uint32_t* SFout) {
  using PackedVec = PackedVec<Type>;
  static constexpr int CVT_FP4_NUM_THREADS_PER_SF =
      (CVT_FP4_SF_VEC_SIZE / CVT_FP4_ELTS_PER_THREAD);
  static_assert(sizeof(PackedVec) == sizeof(Type) * CVT_FP4_ELTS_PER_THREAD,
                "Vec size is not matched.");

  // Get the global scaling factor, which will be applied to the SF.
  // Note SFScale is the same as next GEMM's alpha, which is
  // (448.f / (Alpha_A / 6.f)).
  float const SFScaleVal = SFScale == nullptr ? 1.0f : SFScale[0];

  // Input tensor row/col loops.
  for (int rowIdx = blockIdx.x; rowIdx < numRows; rowIdx += gridDim.x) {
    for (int colIdx = threadIdx.x; colIdx < numCols / CVT_FP4_ELTS_PER_THREAD;
         colIdx += blockDim.x) {
      int64_t inOffset = rowIdx * (numCols / CVT_FP4_ELTS_PER_THREAD) + colIdx;
      PackedVec in_vec = reinterpret_cast<PackedVec const*>(in)[inOffset];
      // Get the output tensor offset.
      // Same as inOffset because 8 elements are packed into one uint32_t.
      int64_t outOffset = inOffset;
      auto& out_pos = out[outOffset];

      auto sf_out =
          cvt_quant_to_fp4_get_sf_out_offset<uint32_t,
                                             CVT_FP4_NUM_THREADS_PER_SF>(
              rowIdx, colIdx, numCols, SFout);

      out_pos =
          cvt_warp_fp16_to_fp4<Type, UE8M0_SF>(in_vec, SFScaleVal, sf_out);
    }
  }
}

}  // namespace vllm
