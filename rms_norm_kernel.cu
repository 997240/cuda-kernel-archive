#include "hip/hip_runtime.h"
#include <torch/hip/hip_runtime.h>
#include <c10/cuda/CUDAGuard.h>

namespace vllm {

template <typename scalar_t>
__global__ void rms_norm_kernel(
    scalar_t* __restrict__ out,          // [..., hidden_size]
    const scalar_t* __restrict__ input,  // [..., hidden_size]
    const int64_t input_stride,
    const scalar_t* __restrict__ weight,  // [hidden_size]
    const float epsilon, const int num_tokens, const int hidden_size) {
  __shared__ float s_variance;
  float variance = 0.0f;
  const scalar_t* input_row = input + blockIdx.x * input_stride;

  constexpr int VEC_SIZE = 8;
  auto vec_op = [&variance](const vec_n_t<scalar_t, VEC_SIZE>& vec) {
#pragma unroll
    for (int i = 0; i < VEC_SIZE; ++i) {
      float x = static_cast<float>(vec.val[i]);
      variance += x * x;
    }
  };
  auto scalar_op = [&variance](const scalar_t& val) {
    float x = static_cast<float>(val);
    variance += x * x;
  };
  vllm::vectorize_read_with_alignment<VEC_SIZE>(
      input_row, hidden_size, threadIdx.x, blockDim.x, vec_op, scalar_op);

  using BlockReduce = hipcub::BlockReduce<float, 1024>;
  __shared__ typename BlockReduce::TempStorage reduceStore;
  variance = BlockReduce(reduceStore).Reduce(variance, CubAddOp{}, blockDim.x);

  if (threadIdx.x == 0) {
    s_variance = rsqrtf(variance / hidden_size + epsilon);
  }
  __syncthreads();

  for (int idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float x = (float)input[blockIdx.x * input_stride + idx];
    out[blockIdx.x * hidden_size + idx] =
        ((scalar_t)(x * s_variance)) * weight[idx];
  }
}

}  // namespace vllm
